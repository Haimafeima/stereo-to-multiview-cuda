#include "hip/hip_runtime.h"
#ifndef D_FILTER_BILATERAL_KERNEL
#define D_FILTER_BILATERAL_KERNEL
#include "d_filter_bilateral.h"
#include "d_filter_gaussian.h"
#include "cuda_utils.h"
#include <math.h>

#define PI 3.14159265359f

inline __device__ float gaussian1D(float x, float sigma)
{
    float variance = pow(sigma, 2);
    float power = pow(x, 2);
    float exponent = -power/(2*variance);
    return __expf(exponent) / sqrt(2 * PI * variance);
}

inline __device__ float gaussian1D_REG(float x, float variance, float sqrt_pi_variance)
{
    float g1d = -(x*x)/(2*variance);
    g1d = __expf(g1d);
    g1d /= sqrt_pi_variance;
    return g1d;
}

float gaussian1D_host(float x, float sigma)
{
    float variance = pow(sigma, 2);
    float power = pow(x, 2);
    float exponent = -power/(2*variance);
    return exp(exponent) / sqrt(2 * PI * variance);
}


void generateGaussian1D(float* kernel, int size, float sigma)
{
    for (int i = 0; i < size; ++i)
        kernel[i] = gaussian1D_host(i, sigma);
}

texture<float, 1, hipReadModeElementType> tex;

__global__ void filter_bilateral_1_kernel_5(float *img_out, float* kernel,
                                            int radius, float sigma_color, float sigma_color_sqrt_pi,
                                            int num_rows, int num_cols,
                                            int sm_img_rows, int sm_img_cols, int sm_img_sz, int sm_img_padding,
                                            int sm_kernel_len, int sm_kernel_sz)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    extern __shared__ float sm_memory[];
    float* sm_img = sm_memory;
    float* sm_kernel = sm_memory + sm_img_sz;

    // Populate Shared Memory IMG
    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_img_rows;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_img_cols;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             
             int sm_idx = tsx + tsy * sm_img_cols;
             int gm_idx = gsx + gsy * num_cols;
             sm_img[sm_idx] = tex1Dfetch(tex, gm_idx);
         }
    }

    // Populate Shared Memory KERNEL

    for (int gsy = ty, tsy = ty;
         tsy < sm_kernel_len;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = tx, tsx = tx; 
              tsx < sm_kernel_len;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_kernel_len;
             int gm_idx = gsx + gsy * sm_kernel_len;

             sm_kernel[sm_idx] = kernel[gm_idx];
         }
    }

    __syncthreads();
    
    float val_a = sm_img[tx + sm_img_padding + (ty + sm_img_padding) * sm_img_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + sm_img_padding + x;
            int sy = ty + sm_img_padding + y;

            float val_s = sm_img[sx + sy * sm_img_cols];
           
            float val_gspatial = sm_kernel[(x + radius) + (y + radius) * kernel_width];
            float val_gcolor = gaussian1D_REG(val_a - val_s, sigma_color, sigma_color_sqrt_pi);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;
    img_out[gx + gy * num_cols] = res;
}
__global__ void filter_bilateral_1_kernel_4(float *img_out, float* kernel,
                                            int radius, float sigma_color, float sigma_color_sqrt_pi,
                                            int num_rows, int num_cols)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;
    
    int idx = gx + gy * num_cols;
    float val_a = tex1Dfetch(tex, idx);

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = gx + x;
            int sy = gy + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = tex1Dfetch(tex, sx + sy * num_cols);

            float val_gspatial = kernel[(x + radius) + (y + radius) * kernel_width];
            float val_gcolor = gaussian1D_REG(val_a - val_s, sigma_color, sigma_color_sqrt_pi);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;

    img_out[gx + gy * num_cols] = res;
}

void filter_bilateral_1_tex(float *img,
                            int radius, float sigma_color, float sigma_spatial,
                            int num_rows, int num_cols)
{
    cudaEventPair_t timer;
	
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int sm_img_rows = bh + 2 * radius;
    int sm_img_cols = bw + 2 * radius;
    int sm_img_sz = sm_img_rows * sm_img_cols;
    int sm_img_padding = radius;

    int sm_kernel_len = 2 * radius + 1;
    int sm_kernel_sz = sm_kernel_len * sm_kernel_len; 
    
    int kernel_sz = sm_kernel_sz; 
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_in;
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    hipBindTexture(0, tex, d_img_in, sizeof(float) * num_rows * num_cols);

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    startCudaTimer(&timer);
    filter_bilateral_1_kernel_5<<<grid_sz, block_sz, sizeof(float) * (sm_img_sz + sm_kernel_sz)>>>(d_img_out, d_kernel, radius, sigma_color, sqrt(2 * PI * sigma_color), num_rows, num_cols, sm_img_rows, sm_img_cols, sm_img_sz, sm_img_padding, sm_kernel_len, sm_kernel_sz);
    stopCudaTimer(&timer, "Bilateral Filter (1 Component) Kernel #5");
    
    checkCudaError(hipMemcpy(img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
    hipFree(d_img_in);
}

__global__ void filter_bilateral_1_kernel_6(float *img_out, float *img_in, 
                                            float* spatial_kernel, float* color_kernel,
                                            int radius, 
                                            int num_rows, int num_cols,
                                            int sm_img_rows, int sm_img_cols, int sm_img_sz, int sm_img_padding,
                                            int sm_spatial_kernel_len, int sm_spatial_kernel_sz,
                                            int sm_color_kernel_len)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    extern __shared__ float sm_memory[];
    float* sm_img = sm_memory;
    float* sm_spatial_kernel = sm_memory + sm_img_sz;
    float* sm_color_kernel = sm_spatial_kernel + sm_spatial_kernel_sz;
    
    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_img_rows;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_img_cols;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_img_cols;
             int gm_idx = min(max(gsx, 0), num_cols - 1) + (min(max(gsy, 0), num_rows - 1) * num_cols);
             sm_img[sm_idx] = img_in[gm_idx];
         }
    }
    
    for (int tsy = ty;
         tsy < sm_spatial_kernel_len;
         tsy += blockDim.y)
    {
         for (int tsx = tx; 
              tsx < sm_spatial_kernel_len;
              tsx += blockDim.x)
         {
             int idx = tsx + tsy * sm_spatial_kernel_len;

             sm_spatial_kernel[idx] = spatial_kernel[idx];
         }
    }
    
    if (ty == 0)
         for (int tsx = tx; tsx < sm_color_kernel_len; tsx += blockDim.x)
             sm_color_kernel[tsx] = color_kernel[tsx];

    __syncthreads();

    float val_a = sm_img[tx + sm_img_padding + (ty + sm_img_padding) * sm_img_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        int sy = ty + sm_img_padding + y;
        int sy_sm_img_cols = sy * sm_img_cols;
        int y_radius_kernel_width =  (y + radius) * kernel_width;
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + sm_img_padding + x;
            float val_s = sm_img[sx + sy_sm_img_cols];
           
            float val_gspatial = sm_spatial_kernel[(x + radius) + y_radius_kernel_width];
            float val_gcolor = sm_color_kernel[(int)abs(val_a - val_s)];
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    img_out[gx + gy * num_cols] = res / norm;
}


__global__ void filter_bilateral_1_kernel_3(float *img_out, float *img_in, float* kernel,
                                            int radius, 
                                            float sigma_color, float sigma_color_sqrt_pi, float sigma_spatial,
                                            int num_rows, int num_cols,
                                            int sm_img_rows, int sm_img_cols, int sm_img_sz, int sm_img_padding,
                                            int sm_kernel_len, int sm_kernel_sz)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    extern __shared__ float sm_memory[];
    float* sm_img = sm_memory;
    float* sm_kernel = sm_memory + sm_img_sz;
    
    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_img_rows;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_img_cols;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_img_cols;
             int gm_idx = min(max(gsx, 0), num_cols - 1) + (min(max(gsy, 0), num_rows - 1) * num_cols);
             sm_img[sm_idx] = img_in[gm_idx];
         }
    }
    
    for (int gsy = ty, tsy = ty;
         tsy < sm_kernel_len;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = tx, tsx = tx; 
              tsx < sm_kernel_len;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_kernel_len;
             int gm_idx = gsx + gsy * sm_kernel_len;

             sm_kernel[sm_idx] = kernel[gm_idx];
         }
    }

    __syncthreads();

    float val_a = sm_img[tx + sm_img_padding + (ty + sm_img_padding) * sm_img_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        int sy = ty + sm_img_padding + y;
        int sy_sm_img_cols = sy * sm_img_cols;
        int y_radius_kernel_width =  (y + radius) * kernel_width;
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + sm_img_padding + x;

            float val_s = sm_img[sx + sy_sm_img_cols];
           
            float val_gspatial = sm_kernel[(x + radius) + y_radius_kernel_width];
            float val_gcolor = gaussian1D_REG(val_a - val_s, sigma_color, sigma_color_sqrt_pi);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;
    img_out[gx + gy * num_cols] = res;
}

__global__ void filter_bilateral_1_kernel_2(float *img_out, float *img_in, float* kernel,
                                            int radius, float sigma_color, float sigma_spatial,
                                            int num_rows, int num_cols,
                                            int sm_img_rows, int sm_img_cols, int sm_img_sz, int sm_img_padding,
                                            int sm_kernel_len, int sm_kernel_sz)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    extern __shared__ float sm_memory[];
    float* sm_img = sm_memory;
    float* sm_kernel = sm_memory + sm_img_sz;
    
    // Populate Shared Memory IMG
    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_img_rows;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_img_cols;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_img_cols;
             
             int gm_idx = min(max(gsx, 0), num_cols - 1) + min(max(gsy, 0), num_rows - 1) * num_cols;

             sm_img[sm_idx] = img_in[gm_idx];
         }
    }

    for (int gsy = ty, tsy = ty;
         tsy < sm_kernel_len;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = tx, tsx = tx; 
              tsx < sm_kernel_len;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_kernel_len;
             int gm_idx = gsx + gsy * sm_kernel_len;

             sm_kernel[sm_idx] = kernel[gm_idx];
         }
    }

    __syncthreads();

    float val_a = sm_img[tx + sm_img_padding + (ty + sm_img_padding) * sm_img_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + sm_img_padding + x;
            int sy = ty + sm_img_padding + y;

            float val_s = sm_img[sx + sy * sm_img_cols];

            float val_gspatial = sm_kernel[(x + radius) + (y + radius) * kernel_width];
            float val_gcolor = gaussian1D(val_a - val_s, sigma_color);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;
    img_out[gx + gy * num_cols] = res;
}


__global__ void filter_bilateral_1_kernel(float *img_out, float *img_in, float* kernel,
                                          int radius, float sigma_color, float sigma_spatial,
                                          int num_rows, int num_cols)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;



    float val_a = img_in[gx + gy * num_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = gx + x;
            int sy = gy + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = img_in[sx + sy * num_cols];

            float val_gspatial = kernel[(x + radius) + (y + radius) * kernel_width];
            float val_gcolor = gaussian1D(val_a - val_s, sigma_color);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;

    img_out[gx + gy * num_cols] = res;
}


void d_filter_bilateral_1(float *d_img,
                          int radius, float sigma_color, float sigma_spatial,
                          int num_rows, int num_cols, int num_disp)
{
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 30;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int sm_img_rows = bh + 2 * radius;
    int sm_img_cols = bw + 2 * radius;
    int sm_img_sz = sm_img_rows * sm_img_cols;
    int sm_img_padding = radius;

    int sm_spatial_kernel_len = 2 * radius + 1;
    int spatial_kernel_sz = sm_spatial_kernel_len * sm_spatial_kernel_len; 
    
    float* spatial_kernel = (float*) malloc(sizeof(float) * spatial_kernel_sz);
    generateGaussianKernel(spatial_kernel, radius, sigma_spatial);

    int color_kernel_sz = num_disp;
    float* color_kernel = (float*) malloc(sizeof(float) * color_kernel_sz);
    generateGaussian1D(color_kernel, color_kernel_sz, sigma_color);
    
    // Device Memory Allocation & Copy
    float* d_img_out;
    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));
    
    float* d_spatial_kernel;
    checkCudaError(hipMalloc(&d_spatial_kernel, sizeof(float) * spatial_kernel_sz));
    checkCudaError(hipMemcpy(d_spatial_kernel, spatial_kernel, sizeof(float) * spatial_kernel_sz, hipMemcpyHostToDevice));
    
    float* d_color_kernel;
    checkCudaError(hipMalloc(&d_color_kernel, sizeof(float) * color_kernel_sz));
    checkCudaError(hipMemcpy(d_color_kernel, color_kernel, sizeof(float) * color_kernel_sz, hipMemcpyHostToDevice));
    
    filter_bilateral_1_kernel_6<<<grid_sz, block_sz, sizeof(float) * (sm_img_sz + spatial_kernel_sz + color_kernel_sz)>>>(d_img_out, d_img, d_spatial_kernel, d_color_kernel, radius, num_rows, num_cols, sm_img_rows, sm_img_cols, sm_img_sz, sm_img_padding, sm_spatial_kernel_len, spatial_kernel_sz, color_kernel_sz);
    hipDeviceSynchronize(); 
    
    checkCudaError(hipMemcpy(d_img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToDevice));
    
    hipFree(d_img_out);
    free(spatial_kernel);
    free(color_kernel);
    hipFree(d_spatial_kernel);
    hipFree(d_color_kernel);
}

void filter_bilateral_1(float *img, 
                        int radius, float sigma_color, float sigma_spatial,
                        int num_rows, int num_cols, int num_disp)
{
    cudaEventPair_t timer;
	
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 30;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int sm_img_rows = bh + 2 * radius;
    int sm_img_cols = bw + 2 * radius;
    int sm_img_sz = sm_img_rows * sm_img_cols;
    int sm_img_padding = radius;

    int sm_spatial_kernel_len = 2 * radius + 1;
    int spatial_kernel_sz = sm_spatial_kernel_len * sm_spatial_kernel_len; 
    
    float* spatial_kernel = (float*) malloc(sizeof(float) * spatial_kernel_sz);
    generateGaussianKernel(spatial_kernel, radius, sigma_spatial);

    int color_kernel_sz = num_disp;
    float* color_kernel = (float*) malloc(sizeof(float) * color_kernel_sz);
    generateGaussian1D(color_kernel, color_kernel_sz, sigma_color);
    
    // Device Memory Allocation & Copy
    float* d_img_in;
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_spatial_kernel;
    checkCudaError(hipMalloc(&d_spatial_kernel, sizeof(float) * spatial_kernel_sz));
    checkCudaError(hipMemcpy(d_spatial_kernel, spatial_kernel, sizeof(float) * spatial_kernel_sz, hipMemcpyHostToDevice));
    
    float* d_color_kernel;
    checkCudaError(hipMalloc(&d_color_kernel, sizeof(float) * color_kernel_sz));
    checkCudaError(hipMemcpy(d_color_kernel, color_kernel, sizeof(float) * color_kernel_sz, hipMemcpyHostToDevice));
    
    startCudaTimer(&timer);
    filter_bilateral_1_kernel_6<<<grid_sz, block_sz, sizeof(float) * (sm_img_sz + spatial_kernel_sz + color_kernel_sz)>>>(d_img_out, d_img_in, d_spatial_kernel, d_color_kernel, radius, num_rows, num_cols, sm_img_rows, sm_img_cols, sm_img_sz, sm_img_padding, sm_spatial_kernel_len, spatial_kernel_sz, color_kernel_sz);
    stopCudaTimer(&timer, "Bilateral Filter (1 Component) Kernel #6");
    
    checkCudaError(hipMemcpy(img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    free(spatial_kernel);
    free(color_kernel);
    hipFree(d_spatial_kernel);
    hipFree(d_color_kernel);
    hipFree(d_img_out);
    hipFree(d_img_in);
}

#endif
