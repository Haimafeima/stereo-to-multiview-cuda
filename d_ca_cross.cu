#include "hip/hip_runtime.h"
#ifndef D_CA_CROSS_KERNEL 
#define D_CA_CROSS_KERNEL
#include "d_ca_cross.h"
#include "cuda_utils.h"
#include <math.h>

#define CROSS_ARM_COUNT 4

typedef enum
{
    CROSS_ARM_UP = 0,
    CROSS_ARM_DOWN,
    CROSS_ARM_LEFT,
    CROSS_ARM_RIGHT
} cross_arm_e;

__global__ void ca_cross_construction_kernel(unsigned char* img, unsigned char** cross,
                                             float ucd, float lcd, int usd, int lsd,
                                             int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    unsigned char a_color_b = img[(tx + ty * num_cols) * elem_sz];
    unsigned char a_color_g = img[(tx + ty * num_cols) * elem_sz + 1];
    unsigned char a_color_r = img[(tx + ty * num_cols) * elem_sz + 2];
    
    // Upper arm
    for (int y = 1; y <= usd; ++y)
    {
        if (ty - y < 0)
            break;
        
        cross[CROSS_ARM_UP][tx + ty * num_cols] = y;
        
        int c_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }

    // Down arm
    for (int y = 1; y <= usd; ++y)
    {
        if (ty + y < 0)
            break;
        
        cross[CROSS_ARM_DOWN][tx + ty * num_cols] = y;
        
        int c_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Left arm
    for (int x = 1; x <= usd; ++x)
    {
        if (tx - x < 0)
            break;
        
        cross[CROSS_ARM_LEFT][tx + ty * num_cols] = x;
        
        int c_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Right arm
    for (int x = 1; x <= usd; ++x)
    {
        if (tx + x < 0)
            break;
        
        cross[CROSS_ARM_RIGHT][tx + ty * num_cols] = x;
        
        int c_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
}

void ca_cross(unsigned char* img_l, unsigned char* img_r, float** cost_l, float** cost_r,
              float** acost_l, float** acost_r, float ucd, float lcd, int usd, int lsd,
              int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    //////////////////////// 
    // CROSS CONSTRUCTION //
    ////////////////////////

    unsigned char* d_img_l;
    unsigned char* d_img_r;

    checkCudaError(hipMalloc(&d_img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    checkCudaError(hipMemcpy(d_img_l, img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_img_r, img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
   
    unsigned char** d_cross_l;
    unsigned char** d_cross_r;
    checkCudaError(hipMalloc(&d_cross_l, sizeof(unsigned char*) * CROSS_ARM_COUNT));
    checkCudaError(hipMalloc(&d_cross_r, sizeof(unsigned char*) * CROSS_ARM_COUNT));

    unsigned char** h_cross_l = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);
    unsigned char** h_cross_r = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);
    
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        checkCudaError(hipMalloc(&h_cross_l[i], sizeof(unsigned char) * num_rows * num_cols));
        checkCudaError(hipMalloc(&h_cross_r[i], sizeof(unsigned char) * num_rows * num_cols));
    }

    checkCudaError(hipMemcpy(d_cross_l, h_cross_l, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_cross_r, h_cross_r, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    
    // Launch kernel
    startCudaTimer(&timer);
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img_l, d_cross_l, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Cross Aggragation - Cross Construciton Kernel");
    
    startCudaTimer(&timer);
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img_r, d_cross_r, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Cross Aggragation - Cross Construciton Kernel");

    ///////////////////
    // DE-ALLOCATION // 
    ///////////////////

    hipFree(d_img_l);
    hipFree(d_img_r);
    hipFree(d_cross_l);
    hipFree(d_cross_r);
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        hipFree(h_cross_l[i]);
        hipFree(h_cross_r[i]);
    }
    free(h_cross_l);
    free(h_cross_r);
}

#endif
