#include "hip/hip_runtime.h"
#ifndef D_CA_CROSS_KERNEL 
#define D_CA_CROSS_KERNEL
#include "d_ca_cross.h"
#include "cuda_utils.h"
#include <math.h>

#define CROSS_ARM_COUNT 4

typedef enum
{
    CROSS_ARM_UP = 0,
    CROSS_ARM_DOWN,
    CROSS_ARM_LEFT,
    CROSS_ARM_RIGHT
} cross_arm_e;

__global__ void ca_cross_hsum_kernel(float** cost, float** acost, unsigned char** cross,
                                     int num_disp, int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    for (int d = 0; d < num_disp; ++d)
    {
        float asum = 0;
        int arm_l = (int) cross[CROSS_ARM_LEFT][tx + ty * num_cols];
        int arm_r = (int) cross[CROSS_ARM_RIGHT][tx + ty * num_cols];
        for (int ax = tx - arm_l; ax < tx + arm_r; ++ax)
        {
            asum = asum + cost[d][ax + ty * num_cols];
        }
        acost[d][tx + ty * num_cols] = asum;
    }
}

__global__ void ca_cross_vsum_kernel(float** cost, float** acost, unsigned char** cross,
                                     int num_disp, int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    for (int d = 0; d < num_disp; ++d)
    {
        float asum = 0;
        int arm_u = (int) cross[CROSS_ARM_UP][tx + ty * num_cols];
        int arm_d = (int) cross[CROSS_ARM_DOWN][tx + ty * num_cols];
        for (int ay = ty - arm_u; ay < ty + arm_d; ++ay)
        {
            asum = asum + cost[d][tx + ay * num_cols];
        }
        acost[d][tx + ty * num_cols] = asum;
    }
}

__global__ void ca_cross_construction_kernel(unsigned char* img, unsigned char** cross,
                                             float ucd, float lcd, int usd, int lsd,
                                             int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    unsigned char a_color_b = img[(tx + ty * num_cols) * elem_sz];
    unsigned char a_color_g = img[(tx + ty * num_cols) * elem_sz + 1];
    unsigned char a_color_r = img[(tx + ty * num_cols) * elem_sz + 2];
    
    cross[CROSS_ARM_UP][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_DOWN][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_LEFT][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_RIGHT][tx + ty * num_cols] = 0;
    
    // Upper arm
    for (int y = 1; y <= usd; ++y)
    {
        if (ty - y < 0)
            break;
        
        cross[CROSS_ARM_UP][tx + ty * num_cols] = (unsigned char) y;
        
        int c_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }

    // Down arm
    for (int y = 1; y <= usd; ++y)
    {
        if ((ty + y) > (num_rows - 1))
            break;
        
        cross[CROSS_ARM_DOWN][tx + ty * num_cols] = (unsigned char) y;
        
        int c_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Left arm
    for (int x = 1; x <= usd; ++x)
    {
        if (tx - x < 0)
            break;
        
        cross[CROSS_ARM_LEFT][tx + ty * num_cols] = (unsigned char) x;
        
        int c_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Right arm
    for (int x = 1; x <= usd; ++x)
    {
        if ((tx + x) > (num_cols - 1))
            break;
        
        cross[CROSS_ARM_RIGHT][tx + ty * num_cols] = (unsigned char) x;
        
        int c_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
}

void d_ca_cross(unsigned char* d_img, float** d_cost, float **h_cost, 
                float** d_acost, float** h_acost,
                float ucd, float lcd, int usd, int lsd,
                int num_disp, int num_rows, int num_cols, int elem_sz)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    //////////////////////// 
    // CROSS CONSTRUCTION //
    ////////////////////////

    unsigned char** d_cross;
    checkCudaError(hipMalloc(&d_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT));

    unsigned char** h_cross = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);
    
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
        checkCudaError(hipMalloc(&h_cross[i], sizeof(unsigned char) * num_rows * num_cols));

    checkCudaError(hipMemcpy(d_cross, h_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    
    // Launch kernel
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img, d_cross, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    hipDeviceSynchronize();
    
    ///////////////////////////
    // CROSS-AGGRAGATE COSTS // 
    ///////////////////////////

    for (int d = 0; d < num_disp; ++d)
        checkCudaError(hipMalloc(&h_acost[d], sizeof(float) * num_rows * num_cols));

    checkCudaError(hipMemcpy(d_acost, h_acost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    // Launch Kernel
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    hipDeviceSynchronize();
    
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    hipDeviceSynchronize();
    
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    hipDeviceSynchronize();
    
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    hipDeviceSynchronize();
    
    ///////////////////
    // DE-ALLOCATION // 
    ///////////////////

    hipFree(d_cross);
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        hipFree(h_cross[i]);
    }
    free(h_cross);
}

void ca_cross(unsigned char* img, float** cost, float** acost,
              float ucd, float lcd, int usd, int lsd,
              int num_disp, int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    //////////////////////// 
    // CROSS CONSTRUCTION //
    ////////////////////////

    unsigned char* d_img;

    checkCudaError(hipMalloc(&d_img, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img, img, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
   
    unsigned char** d_cross;
    checkCudaError(hipMalloc(&d_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT));

    unsigned char** h_cross = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);
    
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        checkCudaError(hipMalloc(&h_cross[i], sizeof(unsigned char) * num_rows * num_cols));
    }

    checkCudaError(hipMemcpy(d_cross, h_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    
    // Launch kernel
    startCudaTimer(&timer);
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img, d_cross, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Cross Aggragation - Cross Construciton Kernel");
    
    ///////////////////////////
    // CROSS-AGGRAGATE COSTS // 
    ///////////////////////////
    float** d_cost;

    checkCudaError(hipMalloc(&d_cost, sizeof(float*) * num_disp));

    float** h_cost = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_cost[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMemcpy(h_cost[d], cost[d], sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    }

    checkCudaError(hipMemcpy(d_cost, h_cost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));

    
    float** d_acost;
    checkCudaError(hipMalloc(&d_acost, sizeof(float*) * num_disp));

    float** h_acost = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_acost[d], sizeof(float) * num_rows * num_cols));
    }

    checkCudaError(hipMemcpy(d_acost, h_acost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    // Launch Kernel

    // Left
    startCudaTimer(&timer);
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Horizontal Sum");
    
    startCudaTimer(&timer);
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Vertical Sum");
    
    
    startCudaTimer(&timer);
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Vertical Sum");
    
    startCudaTimer(&timer);
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Horizontal Sum");
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMemcpy(acost[d], h_cost[d], sizeof(float) * num_cols * num_rows, hipMemcpyDeviceToHost));
    }
    
     ///////////////////
    // DE-ALLOCATION // 
    ///////////////////

    hipFree(d_img);
    hipFree(d_cross);
    hipFree(d_cost);
    hipFree(d_acost);
    for (int d = 0; d < num_disp; ++d)
    {
        hipFree(h_cost[d]);
        hipFree(h_acost[d]);
    }
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        hipFree(h_cross[i]);
    }
    free(h_cost);
    free(h_acost);
    free(h_cross);
}

#endif
