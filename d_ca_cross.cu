#include "hip/hip_runtime.h"
#ifndef D_CA_CROSS_KERNEL 
#define D_CA_CROSS_KERNEL
#include "d_ca_cross.h"
#include "cuda_utils.h"
#include <math.h>

#define CROSS_ARM_COUNT 4

typedef enum
{
    CROSS_ARM_UP = 0,
    CROSS_ARM_DOWN,
    CROSS_ARM_LEFT,
    CROSS_ARM_RIGHT
} cross_arm_e;

__global__ void ca_cross_construction_kernel(unsigned char* img, unsigned char** cross,
                                             float ucd, float lcd, int usd, int lsd,
                                             int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    unsigned char a_color_b = img[(tx + ty * num_cols) * elem_sz];
    unsigned char a_color_g = img[(tx + ty * num_cols) * elem_sz + 1];
    unsigned char a_color_r = img[(tx + ty * num_cols) * elem_sz + 2];
    
    cross[CROSS_ARM_UP][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_DOWN][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_LEFT][tx + ty * num_cols] = 0;
    cross[CROSS_ARM_RIGHT][tx + ty * num_cols] = 0;
    
    // Upper arm
    for (int y = 1; y <= usd; ++y)
    {
        if (ty - y < 0)
            break;
        
        cross[CROSS_ARM_UP][tx + ty * num_cols] = (unsigned char) y;
        
        int c_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty - y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty - y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }

    // Down arm
    for (int y = 1; y <= usd; ++y)
    {
        if ((ty + y) > (num_rows - 1))
            break;
        
        cross[CROSS_ARM_DOWN][tx + ty * num_cols] = (unsigned char) y;
        
        int c_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + (ty + y) * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + (ty + y) * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (y > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Left arm
    for (int x = 1; x <= usd; ++x)
    {
        if (tx - x < 0)
            break;
        
        cross[CROSS_ARM_LEFT][tx + ty * num_cols] = (unsigned char) x;
        
        int c_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx - x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx - x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx - x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
    
    // Right arm
    for (int x = 1; x <= usd; ++x)
    {
        if ((tx + x) > (num_cols - 1))
            break;
        
        cross[CROSS_ARM_RIGHT][tx + ty * num_cols] = (unsigned char) x;
        
        int c_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int c_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int c_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];
        
        int p_color_b = (int) img[(tx + x + ty * num_cols) * elem_sz];
        int p_color_g = (int) img[(tx + x + ty * num_cols) * elem_sz + 1];
        int p_color_r = (int) img[(tx + x + ty * num_cols) * elem_sz + 2];

        int ac_mad = max(max(abs(c_color_b - a_color_b), abs(c_color_g - a_color_g)), abs(c_color_r - a_color_r));
        int cp_mad = max(max(abs(c_color_b - p_color_b), abs(c_color_g - p_color_g)), abs(c_color_r - p_color_r));

        if (x > lsd)
        {
            if ((float) ac_mad > ucd)
                break;
        }
        else
        {
            if ((float) ac_mad > lcd || (float) cp_mad > lcd)
                break;
        }
    }
}

void d_ca_cross(unsigned char* d_img, float** d_cost, float **h_cost, 
                float** d_acost, float** h_acost, float *d_acost_memory,
                float ucd, float lcd, int usd, int lsd,
                int num_disp, int num_rows, int num_cols, int elem_sz)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 160;
    size_t bh = 1;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    size_t img_sz = num_rows * num_cols;
    size_t imgelem_sz = img_sz * elem_sz;
    size_t cost_sz = img_sz * num_disp;
    
    //////////////////////// 
    // CROSS CONSTRUCTION //
    ////////////////////////

    unsigned char** d_cross;
    checkCudaError(hipMalloc(&d_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT));

    unsigned char** h_cross = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);

    unsigned char* d_cross_memory;
    checkCudaError(hipMalloc(&d_cross_memory, sizeof(unsigned char) * img_sz * CROSS_ARM_COUNT));
    
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
        h_cross[i] = d_cross_memory + (i * img_sz);

    checkCudaError(hipMemcpy(d_cross, h_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img, d_cross, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    hipDeviceSynchronize();
    
    ///////////////////////////
    // CROSS-AGGRAGATE COSTS // 
    ///////////////////////////
	int sm_cols = bw + (usd) * 2;
	int sm_sz = sm_cols * bh;
	int sm_padding = usd;
	
	size_t bw_v = 1;
    size_t bh_v = num_rows / 4;
    size_t gw_v = (num_cols + bw_v - 1) / bw_v;
    size_t gh_v = (num_rows + bh_v - 1) / bh_v;
    const dim3 block_sz_v(bw_v, bh_v, 1);
    const dim3 grid_sz_v(gw_v, gh_v, 1);
	
	int sm_rows = bh_v + (usd) * 2;
	int sm_sz_v = sm_rows * bw_v;
	int sm_padding_v = usd;

    for (int d = 0; d < num_disp; ++d)
        h_acost[d] = d_acost_memory + (d * img_sz);

    checkCudaError(hipMemcpy(d_acost, h_acost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    ca_cross_hsum_kernel_2<<<grid_sz, block_sz, sizeof(float) * sm_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols, sm_cols, sm_sz, sm_padding); 
    hipDeviceSynchronize();
    
    ca_cross_vsum_kernel_2<<<grid_sz_v, block_sz_v, sizeof(float) * sm_sz_v>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols, sm_rows, sm_sz_v, sm_padding_v); 
    hipDeviceSynchronize();
    
    ca_cross_vsum_kernel_2<<<grid_sz_v, block_sz_v, sizeof(float) * sm_sz_v>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols, sm_rows, sm_sz_v, sm_padding_v); 
    hipDeviceSynchronize();
    
    ca_cross_hsum_kernel_2<<<grid_sz, block_sz, sizeof(float) * sm_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols, sm_cols, sm_sz, sm_padding); 
    hipDeviceSynchronize();
    
    
    ///////////////////
    // DE-ALLOCATION // 
    ///////////////////
    
    hipFree(d_cross_memory);
    hipFree(d_cross);
    free(h_cross);
}

void ca_cross(unsigned char* img, float** cost, float** acost,
              float ucd, float lcd, int usd, int lsd,
              int num_disp, int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 160;
    size_t bh = 1;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    
    //////////////////////// 
    // CROSS CONSTRUCTION //
    ////////////////////////

    unsigned char* d_img;

    checkCudaError(hipMalloc(&d_img, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img, img, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
   
    unsigned char** d_cross;
    checkCudaError(hipMalloc(&d_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT));

    unsigned char** h_cross = (unsigned char**) malloc(sizeof(unsigned char*) * CROSS_ARM_COUNT);
    
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        checkCudaError(hipMalloc(&h_cross[i], sizeof(unsigned char) * num_rows * num_cols));
    }

    checkCudaError(hipMemcpy(d_cross, h_cross, sizeof(unsigned char*) * CROSS_ARM_COUNT, hipMemcpyHostToDevice));
    
    // Launch kernel
    startCudaTimer(&timer);
    ca_cross_construction_kernel<<<grid_sz, block_sz>>>(d_img, d_cross, ucd, lcd, usd, lsd, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Cross Aggragation - Cross Construciton Kernel");
    
    ///////////////////////////
    // CROSS-AGGRAGATE COSTS // 
    ///////////////////////////
    float** d_cost;

    checkCudaError(hipMalloc(&d_cost, sizeof(float*) * num_disp));

    float** h_cost = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_cost[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMemcpy(h_cost[d], cost[d], sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    }

    checkCudaError(hipMemcpy(d_cost, h_cost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));

    
    float** d_acost;
    checkCudaError(hipMalloc(&d_acost, sizeof(float*) * num_disp));

    float** h_acost = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_acost[d], sizeof(float) * num_rows * num_cols));
    }

    checkCudaError(hipMemcpy(d_acost, h_acost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
	int sm_cols = bw + (usd) * 2;
	int sm_sz = sm_cols * bh;
	int sm_padding = usd;
	
	size_t bw_v = 1;
    size_t bh_v = num_rows / 2;
    size_t gw_v = (num_cols + bw_v - 1) / bw_v;
    size_t gh_v = (num_rows + bh_v - 1) / bh_v;
    const dim3 block_sz_v(bw_v, bh_v, 1);
    const dim3 grid_sz_v(gw_v, gh_v, 1);
	
	int sm_rows = bh_v + (usd) * 2;
	int sm_sz_v = sm_rows * bw_v;
	int sm_padding_v = usd;

	/*	
	startCudaTimer(&timer);
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Horizontal Sum");
*/
	startCudaTimer(&timer);
    ca_cross_hsum_kernel_2<<<grid_sz, block_sz, sizeof(float) * sm_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols, sm_cols, sm_sz, sm_padding); 
    stopCudaTimer(&timer, "Cross Horizontal Sum #2");
    
	startCudaTimer(&timer);
    ca_cross_vsum_kernel_2<<<grid_sz_v, block_sz_v, sizeof(float) * sm_sz_v>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols, sm_rows, sm_sz_v, sm_padding_v); 
    stopCudaTimer(&timer, "Cross Vertical Sum #2");
    
	startCudaTimer(&timer);
    ca_cross_vsum_kernel_2<<<grid_sz_v, block_sz_v, sizeof(float) * sm_sz_v>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols, sm_rows, sm_sz_v, sm_padding_v); 
    stopCudaTimer(&timer, "Cross Vertical Sum #2");
/*	
	startCudaTimer(&timer);
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Vertical Sum");
    
	startCudaTimer(&timer);
    ca_cross_vsum_kernel<<<grid_sz, block_sz>>>(d_cost, d_acost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Vertical Sum");
*/
	/*
	startCudaTimer(&timer);
    ca_cross_hsum_kernel<<<grid_sz, block_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols); 
    stopCudaTimer(&timer, "Cross Horizontal Sum");
*/	
	startCudaTimer(&timer);
    ca_cross_hsum_kernel_2<<<grid_sz, block_sz, sizeof(float) * sm_sz>>>(d_acost, d_cost, d_cross, num_disp, num_rows, num_cols, sm_cols, sm_sz, sm_padding); 
    stopCudaTimer(&timer, "Cross Horizontal Sum #2");
    
	
	for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMemcpy(acost[d], h_cost[d], sizeof(float) * num_cols * num_rows, hipMemcpyDeviceToHost));
    }
    
     ///////////////////
    // DE-ALLOCATION // 
    ///////////////////

    hipFree(d_img);
    hipFree(d_cross);
    hipFree(d_cost);
    hipFree(d_acost);
    for (int d = 0; d < num_disp; ++d)
    {
        hipFree(h_cost[d]);
        hipFree(h_acost[d]);
    }
    for (int i = 0; i < CROSS_ARM_COUNT; ++i)
    {
        hipFree(h_cross[i]);
    }
    free(h_cost);
    free(h_acost);
    free(h_cross);
}

#endif
