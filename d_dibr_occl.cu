#include "hip/hip_runtime.h"
#ifndef D_DIBR_OCCL_KERNEL 
#define D_DIBR_OCCL_KERNEL
#include "d_dibr_occl.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void dibr_smooth_mask_kernel(float *mask, float *disp,
                                        int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;


}

__global__ void dibr_occl_to_mask_kernel(float *mask, unsigned char *occl,
                                         int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    unsigned char val_occl = occl[tx + ty * num_cols];
    if (val_occl == 1) 
        mask[tx + ty * num_cols] = 1.0f;
    else 
        mask[tx + ty * num_cols] = 0.0f;
}

void d_dibr_occl_to_mask(float *d_mask_l, float *d_mask_r,
                         unsigned char* d_occl_l, unsigned char* d_occl_r,
                         int num_rows, int num_cols)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////////// 
    // KERNEL LAUNCH //
    ///////////////////
    
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_l, d_occl_l, num_rows, num_cols);
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_r, d_occl_r, num_rows, num_cols);
    hipDeviceSynchronize(); 
}


void dibr_occl_to_mask(float *mask_l, float *mask_r,
                       unsigned char* occl_l, unsigned char* occl_r,
                       int num_rows, int num_cols)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////////////// 
    // MEMORY ALLOCATION //
    ///////////////////////
    
    float* d_mask_l, *d_mask_r;

    checkCudaError(hipMalloc(&d_mask_l, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_mask_r, sizeof(float) * num_rows * num_cols));
    
    unsigned char* d_occl_l, *d_occl_r; 

    checkCudaError(hipMalloc(&d_occl_l, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_occl_r, sizeof(unsigned char) * num_rows * num_cols));
    
    checkCudaError(hipMemcpy(d_occl_l, occl_l, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_occl_r, occl_r, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyHostToDevice));

    
    /////////////////// 
    // KERNEL LAUNCH //
    ///////////////////
    
    startCudaTimer(&timer);
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_l, d_occl_l, num_rows, num_cols);
    stopCudaTimer(&timer, "Dis-occlusion Kernel");
    
    startCudaTimer(&timer);
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_r, d_occl_r, num_rows, num_cols);
    stopCudaTimer(&timer, "Dis-occlusion Kernel");

    checkCudaError(hipMemcpy(mask_l, d_mask_l, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(mask_r, d_mask_r, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    hipFree(d_occl_l);
    hipFree(d_occl_r);
    hipFree(d_mask_l);
    hipFree(d_mask_r);
}

__global__ void dibr_find_occlusion_kernel(unsigned char *occl, float *disp,
                                           int dir,
                                           int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    int sd = (disp[tx + ty * num_cols] * dir); 
    int sx = min(max(tx + sd, 0), num_cols - 1);
    
    occl[sx + ty * num_cols] = 1;
}

void d_dibr_occl(unsigned char* d_occl_l, unsigned char* d_occl_r,
                 float* d_disp_l, float* d_disp_r,
                 int num_rows, int num_cols)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////////////// 
    // MEMORY ALLOCATION //
    ///////////////////////
    
    checkCudaError(hipMemset(d_occl_l, 0, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMemset(d_occl_r, 0, sizeof(unsigned char) * num_rows * num_cols));

    /////////////////// 
    // KERNEL LAUNCH //
    ///////////////////
    
    dibr_find_occlusion_kernel<<<grid_sz, block_sz>>>(d_occl_r, d_disp_l, 1, num_rows, num_cols);
    dibr_find_occlusion_kernel<<<grid_sz, block_sz>>>(d_occl_l, d_disp_r, -1, num_rows, num_cols);
    hipDeviceSynchronize(); 
}


void dibr_occl(unsigned char* occl_l, unsigned char* occl_r,
               float* disp_l, float* disp_r,
               int num_rows, int num_cols)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////////////// 
    // MEMORY ALLOCATION //
    ///////////////////////
    
    unsigned char* d_occl_l, *d_occl_r; 

    checkCudaError(hipMalloc(&d_occl_l, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_occl_r, sizeof(unsigned char) * num_rows * num_cols));
    
    checkCudaError(hipMemset(d_occl_l, 0, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMemset(d_occl_r, 0, sizeof(unsigned char) * num_rows * num_cols));

    float* d_disp_l, *d_disp_r;

    checkCudaError(hipMalloc(&d_disp_l, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_disp_r, sizeof(float) * num_rows * num_cols));

    checkCudaError(hipMemcpy(d_disp_l, disp_l, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_disp_r, disp_r, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    
    /////////////////// 
    // KERNEL LAUNCH //
    ///////////////////
    
    startCudaTimer(&timer);
    dibr_find_occlusion_kernel<<<grid_sz, block_sz>>>(d_occl_r, d_disp_l, 1, num_rows, num_cols);
    stopCudaTimer(&timer, "Dis-occlusion Kernel");
    
    startCudaTimer(&timer);
    dibr_find_occlusion_kernel<<<grid_sz, block_sz>>>(d_occl_l, d_disp_r, -1, num_rows, num_cols);
    stopCudaTimer(&timer, "Dis-occlusion Kernel");

    checkCudaError(hipMemcpy(occl_l, d_occl_l, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(occl_r, d_occl_r, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyDeviceToHost));

    hipFree(d_occl_l);
    hipFree(d_occl_r);
    hipFree(d_disp_l);
    hipFree(d_disp_r);
}

#endif
