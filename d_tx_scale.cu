#include "hip/hip_runtime.h"
#ifndef D_TX_SCALE_KERNEL
#define D_TX_SCALE_KERNEL
#include "d_tx_scale.h"
#include "d_alu.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void tx_disp_scale_kernel(float *disp_out, float *disp_in,  
                                     int out_rows, int out_cols, int in_rows, int in_cols,
                                     float disp_scale)
{
    // Thread Id's
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= out_cols || ty >= out_rows)
        return;
    
    // Compute Input Sampling Coordinates
    float x_samp = fmin(fmax(((float) tx / (float) out_cols) * (float) in_cols, 0), (float) (in_cols - 1));
    float y_samp = fmin(fmax(((float) ty / (float) out_rows) * (float) in_rows, 0), (float) (in_rows - 1));
    
    // Write to Output
    int d_out = tx + ty * out_cols;

    disp_out[d_out] = alu_bilinear_interp_f(disp_in, x_samp, y_samp, in_cols, in_rows) * disp_scale;
}


__global__ void tx_scale_bilinear_kernel(unsigned char* img_in, unsigned char* img_out,  
                                         int in_rows, int in_cols, int out_rows, int out_cols, int elem_sz)
{
    // Thread Id's
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if (gx >= out_cols || gy >= out_rows)
        return;
    
    // Compute Input Sampling Coordinates
    float g_x_samp = fmin(fmax(((float) gx / (float) out_cols) * (float) in_cols, 0), (float) (in_cols - 1));
    float g_y_samp = fmin(fmax(((float) gy / (float) out_rows) * (float) in_rows, 0), (float) (in_rows - 1));
    
    // Write to Output
    int b_out = (gx + gy * out_cols) * elem_sz;
    int g_out = b_out + 1;
    int r_out = g_out + 1;

    img_out[b_out] = alu_bilinear_interp(img_in, elem_sz, 0, g_x_samp, g_y_samp, in_cols, in_rows);
    img_out[g_out] = alu_bilinear_interp(img_in, elem_sz, 1, g_x_samp, g_y_samp, in_cols, in_rows);
    img_out[r_out] = alu_bilinear_interp(img_in, elem_sz, 2, g_x_samp, g_y_samp, in_cols, in_rows);
}

__global__ void tx_scale_nearest_kernel(unsigned char* img_in, unsigned char* img_out,  
                                        int in_rows, int in_cols, int out_rows, int out_cols, int elem_sz)
{
    // Thread Id's
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= out_cols || ty >= out_rows)
        return;
    
    // Compute Input Sampling Coordinates
    float x_samp = fmin(fmax(((float) tx / (float) out_cols) * (float) in_cols, 0), (float) (in_cols - 1));
    float y_samp = fmin(fmax(((float) ty / (float) out_rows) * (float) in_rows, 0), (float) (in_rows - 1));
    
    int b_in = ((int) x_samp + (int) y_samp * in_cols) * elem_sz;
    int g_in = b_in + 1;
    int r_in = g_in + 1;

    // Write to Output
    int b_out = (tx + ty * out_cols) * elem_sz;
    int g_out = b_out + 1;
    int r_out = g_out + 1;

    img_out[b_out] = img_in[b_in];
    img_out[g_out] = img_in[g_in];
    img_out[r_out] = img_in[r_in];
}

void d_tx_scale(unsigned char* img_in, unsigned char* img_out, 
                int in_rows, int in_cols, int out_rows, int out_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    // Device Memory Allocation & Copy Data Host -> Device
    unsigned char* d_img_in;
    unsigned char* d_img_out;
    
    checkCudaError(hipMalloc(&d_img_in, sizeof(unsigned char) * in_rows * in_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img_in, img_in, 
                   sizeof(unsigned char) * in_rows * in_cols * elem_sz, hipMemcpyHostToDevice));
    
    checkCudaError(hipMalloc(&d_img_out, sizeof(unsigned char) * out_rows * out_cols * elem_sz));

    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (out_cols + bw - 1) / bw;
    size_t gh = (out_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    // Launch Kernel
    startCudaTimer(&timer);
    tx_scale_bilinear_kernel<<<grid_sz, block_sz>>>(d_img_in, d_img_out,
                                                    in_rows, in_cols, out_rows, out_cols, elem_sz);
                                                               
    stopCudaTimer(&timer, "Scale Kernel - Bilinear"); 
    
    // Copy Data Device -> Host
    checkCudaError(hipMemcpy(img_out, d_img_out, 
                   sizeof(unsigned char) * out_rows * out_cols * elem_sz, hipMemcpyDeviceToHost));

    // Device Memory De-allocation
    hipFree(d_img_in);
    hipFree(d_img_out);
}

#endif
