#include "hip/hip_runtime.h"
#ifndef D_DC_WTA_KERNEL 
#define D_DC_WTA_KERNEL
#include "d_dc_wta.h"
#include "cuda_utils.h"
#include <math.h>
#include <float.h>
#include <limits.h>

__global__ void dc_wta_kernel(float** cost, float* disp, 
                              int num_disp, int zero_disp, 
                              int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    float lowest_cost = FLT_MAX;

    int tx_ty_num_cols = tx + ty * num_cols;

    float lowest_d = 0;
    for (int d = 0; d < num_disp; ++d)
    {
       float current_cost = cost[d][tx_ty_num_cols];
       
       if (lowest_cost > current_cost)
       {
           lowest_cost = current_cost;
           lowest_d = d;
       }
    }
    disp[tx_ty_num_cols] = lowest_d - (float) zero_disp;
}

void d_dc_wta(float** d_cost, float* d_disp, 
              int num_disp, int zero_disp, 
              int num_rows, int num_cols)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////

    size_t bw = 640;
    size_t bh = 1;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    //////////////////////
    // WINNER TAKES ALL //
    //////////////////////
    
    dc_wta_kernel<<<grid_sz, block_sz>>>(d_cost, d_disp, num_disp, zero_disp, num_rows, num_cols);
    hipDeviceSynchronize(); 

}


void dc_wta(float** cost, float* disp, 
            int num_disp, int zero_disp, 
            int num_rows, int num_cols)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////

    size_t bw = 640;
    size_t bh = 1;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    ///////////////////////
    // MEMORY ALLOCATION //
    ///////////////////////

    float** d_cost;

    checkCudaError(hipMalloc(&d_cost, sizeof(float*) * num_disp));

    float** h_cost = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_cost[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMemcpy(h_cost[d], cost[d], sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    }

    checkCudaError(hipMemcpy(d_cost, h_cost, sizeof(float*) * num_disp, hipMemcpyHostToDevice));

    float* d_disp;
    checkCudaError(hipMalloc(&d_disp, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemset(d_disp, 0, sizeof(float) * num_rows * num_cols));
    
    //////////////////////
    // WINNER TAKES ALL //
    //////////////////////
    
    startCudaTimer(&timer); 
    dc_wta_kernel<<<grid_sz, block_sz>>>(d_cost, d_disp, num_disp, zero_disp, num_rows, num_cols);
    stopCudaTimer(&timer, "Disparity Computation Kernel"); 
    
    checkCudaError(hipMemcpy(disp, d_disp, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
    
    ///////////////////
    // DE-ALLOCATION //
    ///////////////////
    
    hipFree(d_cost);
    hipFree(d_disp);
    for (int d = 0; d < num_disp; ++d)
        hipFree(h_cost[d]);
    free(h_cost);
}

#endif
