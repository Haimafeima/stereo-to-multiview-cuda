#include "hip/hip_runtime.h"
#ifndef D_CI_AD_KERNEL 
#define D_CI_AD_KERNEL
#include "d_ci_ad.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void ci_ad_kernel(unsigned char* img_l, unsigned char* img_r, float** cost_l, 
                            float** cost_R, int num_disp, int zero_disp, int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    for (int d = 0; d < num_disp; ++d)
    {
        int r_coord = min(max(tx + (d - zero_disp), 0), num_cols - 1);
        int ll = (tx + ty * num_cols) * elem_sz;
        int lr = (r_coord + ty * num_cols) * elem_sz;
        float cost_b = (float) abs(img_l[ll] - img_r[lr]);
        float cost_g = (float) abs(img_l[ll + 1] - img_r[lr + 1]);
        float cost_r = (float) abs(img_l[ll + 2] - img_r[lr + 2]);
        float cost = (cost_b + cost_g + cost_r) / 3.0;
        cost_l[d][tx + ty * num_cols] = cost;
        cost_R[d][r_coord + ty * num_cols] = cost;
    }
}

void ci_ad(unsigned char* img_l, unsigned char* img_r, float** cost_l, float** cost_r, int num_disp, int zero_disp, int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    // Device Memory Allocation & Copy
    unsigned char* d_img_l;
    unsigned char* d_img_r;

    checkCudaError(hipMalloc(&d_img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img_l, img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img_r, img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));

    // Device Cost Memory
    float** d_cost_l;
    float** d_cost_r;
    
    checkCudaError(hipMalloc(&d_cost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_cost_r, sizeof(float*) * num_disp));
    
    float** h_cost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_cost_r = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_cost_l[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMalloc(&h_cost_r[d], sizeof(float) * num_rows * num_cols));
    }
    
    checkCudaError(hipMemcpy(d_cost_l, h_cost_l, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_cost_r, h_cost_r, sizeof(float*) * num_disp, hipMemcpyHostToDevice));

	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    // Launch Kernel
    startCudaTimer(&timer);
    ci_ad_kernel<<<grid_sz, block_sz>>>(d_img_l, d_img_r, d_cost_l, d_cost_r, num_disp, zero_disp, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Cost Initialization - Absolute Difference Kernel");
    
    // Copy Device Data to Host
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMemcpy(cost_l[d], h_cost_l[d], sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
        checkCudaError(hipMemcpy(cost_r[d], h_cost_r[d], sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
    }

    
    // Deallocation
    hipFree(d_img_l);
    hipFree(d_img_r);
    hipFree(d_cost_l);
    hipFree(d_cost_r);
    for (int d = 0; d < num_disp; ++d)
    {
        hipFree(h_cost_l[d]);
        hipFree(h_cost_r[d]);
    }
}

#endif
