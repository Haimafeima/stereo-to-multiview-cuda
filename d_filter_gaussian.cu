#include "hip/hip_runtime.h"
#ifndef D_FILTER_GAUSSIAN_KERNEL
#define D_FILTER_GAUSSIAN_KERNEL
#include "d_filter_gaussian.h"
#include "cuda_utils.h"
#include <math.h>

#define PI 3.14159265359f

__global__ void filter_gaussian_1_kernel_1(float* img_out, float* img_in,
                                          float *kernel,
                                          int radius, float sigma_spatial,
                                          int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    float val_a = img_in[tx + ty * num_cols];
    int kernel_width = radius * 2 + 1;
    float res = 0.0f;
    float norm = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + x;
            int sy = ty + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = img_in[sx + sy * num_cols];
            float weight = kernel[(x + radius) + (y + radius) * kernel_width];
            
            norm = norm + weight;
            res = res + (val_s * weight); 
            
        }
    }
    if (val_a < res/norm)
        img_out[tx + ty * num_cols] = res/norm;
    else
        img_out[tx + ty * num_cols] = val_a;
}


__global__ void filter_gaussian_1_kernel(float* img_out, float* img_in,
                                          float *kernel,
                                          int radius, float sigma_spatial,
                                          int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    float val_a = img_in[tx + ty * num_cols];
    int kernel_width = radius * 2 + 1;
    float res = 0.0f;
    float norm = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + x;
            int sy = ty + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = img_in[sx + sy * num_cols];
            float weight = kernel[(x + radius) + (y + radius) * kernel_width];
            
            norm = norm + weight;
            res = res + (val_s * weight); 
            
        }
    }
    if (val_a < res/norm)
        img_out[tx + ty * num_cols] = res/norm;
    else
        img_out[tx + ty * num_cols] = val_a;
}

void d_filter_gaussian_1(float *d_img,
                          int radius, float sigma_spatial,
                          int num_rows, int num_cols)
{
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    filter_gaussian_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img, d_kernel, radius, sigma_spatial, num_rows, num_cols);
    hipDeviceSynchronize();
    
    checkCudaError(hipMemcpy(d_img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToDevice));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
}


void filter_gaussian_1(float *img,
                        int radius, float sigma_spatial,
                        int num_rows, int num_cols)
{
    cudaEventPair_t timer;
	
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_in;
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    startCudaTimer(&timer);
    filter_gaussian_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in, d_kernel, radius, sigma_spatial, num_rows, num_cols);
    stopCudaTimer(&timer, "Gaussian Filter (1 FLoat Component) Kernel");
    
    checkCudaError(hipMemcpy(img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
    hipFree(d_img_in);
}

__host__ __device__ float gaussian2D(float x, float y, float sigma)
{
    float variance = pow(sigma,2);
    float exponent = -(pow(x,2) + pow(y,2))/(2 * variance);
    return expf(exponent) / (2 * PI * variance);
}

void generateGaussianKernel(float* kernel, int radius, float sigma)
{
    int kernel_width = radius * 2 + 1;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            kernel[(x + radius) + (y + radius) * kernel_width] = gaussian2D(x, y, sigma);
        }
    }
}

#endif
