#include "hip/hip_runtime.h"
#ifndef D_FILTER_GAUSSIAN_KERNEL
#define D_FILTER_GAUSSIAN_KERNEL
#include "d_filter_gaussian.h"
#include "cuda_utils.h"
#include <math.h>

#define PI 3.14159265359f

__global__ void filter_gaussian_1_kernel_1(float* img_out, float* img_in,
                                          float *kernel,
                                          int radius, float sigma_spatial,
                                          int num_rows, int num_cols,
                                          int sm_img_rows, int sm_img_cols, int sm_img_sz, int sm_img_padding,
                                          int sm_kernel_len, int sm_kernel_sz)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    extern __shared__ float sm_memory[];
    float* sm_img = sm_memory;
    float* sm_kernel = sm_memory + sm_img_sz;
    
    // Populate Shared Memory IMG
    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_img_rows;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_img_cols;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_img_cols;
             int gm_idx = min(max(gsx, 0), num_cols - 1) + min(max(gsy, 0), num_rows - 1) * num_cols;

             sm_img[sm_idx] = img_in[gm_idx];
         }
    }

    for (int gsy = gy - sm_img_padding, tsy = ty;
         tsy < sm_kernel_len;
         gsy += blockDim.y, tsy += blockDim.y)
    {
         for (int gsx = gx - sm_img_padding, tsx = tx; 
              tsx < sm_kernel_len;
              gsx += blockDim.x, tsx += blockDim.x)
         {
             int sm_idx = tsx + tsy * sm_kernel_len;
             int gm_idx = min(max(gsx, 0), num_cols - 1) + min(max(gsy, 0), num_rows - 1) * num_cols;

             sm_kernel[sm_idx] = kernel[gm_idx];
         }
    }

    __syncthreads();

    
    float val_a = sm_img[tx + sm_img_padding + (ty + sm_img_padding) * sm_img_cols];
    
    int kernel_width = radius * 2 + 1;
    float res = 0.0f;
    float norm = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + sm_img_padding + x;
            int sy = ty + sm_img_padding + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = sm_img[sx + sy * sm_img_cols];
            float weight = sm_kernel[(x + radius) + (y + radius) * kernel_width];
            
            norm = norm + weight;
            res = res + (val_s * weight); 
            
        }
    }
    if (val_a < res/norm)
        img_out[gx + gy * num_cols] = res/norm;
    else
        img_out[gx + gy * num_cols] = val_a;
}



__global__ void filter_gaussian_1_kernel(float* img_out, float* img_in,
                                          float *kernel,
                                          int radius, float sigma_spatial,
                                          int num_rows, int num_cols)
{
    int gx = threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((gx > num_cols - 1) || (gy > num_rows - 1))
        return;
    
    float val_a = img_in[gx + gy * num_cols];
    int kernel_width = radius * 2 + 1;
    float res = 0.0f;
    float norm = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = gx + x;
            int sy = gy + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = img_in[sx + sy * num_cols];
            float weight = kernel[(x + radius) + (y + radius) * kernel_width];
            
            norm = norm + weight;
            res = res + (val_s * weight); 
            
        }
    }
    if (val_a < res/norm)
        img_out[gx + gy * num_cols] = res/norm;
    else
        img_out[gx + gy * num_cols] = val_a;
}

void d_filter_gaussian_1(float *d_img,
                          int radius, float sigma_spatial,
                          int num_rows, int num_cols)
{
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int sm_img_rows = bh + 2 * radius;
    int sm_img_cols = bw + 2 * radius;
    int sm_img_sz = sm_img_rows * sm_img_cols;
    int sm_img_padding = radius;

    int sm_kernel_len = 2 * radius + 1;
    int sm_kernel_sz = sm_kernel_len * sm_kernel_len; 
    
    int kernel_sz = sm_kernel_sz; 
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    filter_gaussian_1_kernel_1<<<grid_sz, block_sz, sizeof(float) * (sm_img_sz + sm_kernel_sz)>>>(d_img_out, d_img, d_kernel, radius, sigma_spatial, num_rows, num_cols, sm_img_rows, sm_img_cols, sm_img_sz, sm_img_padding, sm_kernel_len, sm_kernel_sz);
    hipDeviceSynchronize();
    
    checkCudaError(hipMemcpy(d_img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToDevice));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
}


void filter_gaussian_1(float *img,
                        int radius, float sigma_spatial,
                        int num_rows, int num_cols)
{
    cudaEventPair_t timer;
	
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int sm_img_rows = bh + 2 * radius;
    int sm_img_cols = bw + 2 * radius;
    int sm_img_sz = sm_img_rows * sm_img_cols;
    int sm_img_padding = radius;

    int sm_kernel_len = 2 * radius + 1;
    int sm_kernel_sz = sm_kernel_len * sm_kernel_len; 
    
    int kernel_sz = sm_kernel_sz; 
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_in;
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    startCudaTimer(&timer);
    filter_gaussian_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in, d_kernel, radius, sigma_spatial, num_rows, num_cols);
    stopCudaTimer(&timer, "Gaussian Filter (1 FLoat Component) Kernel");
    
    startCudaTimer(&timer);
    filter_gaussian_1_kernel_1<<<grid_sz, block_sz, sizeof(float) * (sm_img_sz + sm_kernel_sz)>>>(d_img_out, d_img_in, d_kernel, radius, sigma_spatial, num_rows, num_cols, sm_img_rows, sm_img_cols, sm_img_sz, sm_img_padding, sm_kernel_len, sm_kernel_sz);
    stopCudaTimer(&timer, "Gaussian Filter (1 FLoat Component) Kernel");
    
    checkCudaError(hipMemcpy(img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
    hipFree(d_img_in);
}

__host__ __device__ float gaussian2D(float x, float y, float sigma)
{
    float variance = pow(sigma,2);
    float exponent = -(pow(x,2) + pow(y,2))/(2 * variance);
    return expf(exponent) / (2 * PI * variance);
}

void generateGaussianKernel(float* kernel, int radius, float sigma)
{
    int kernel_width = radius * 2 + 1;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            kernel[(x + radius) + (y + radius) * kernel_width] = gaussian2D(x, y, sigma);
        }
    }
}

#endif
