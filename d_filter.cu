#include "hip/hip_runtime.h"
#ifndef D_FILTER_KERNEL
#define D_FILTER_KERNEL
#include "d_filter.h"
#include "cuda_utils.h"
#include <math.h>

#define PI 3.14159265359f

__global__ void filter_gauss_1F_kernel(float* img_out, float* img_in,
                                       float sigma_spatial, int radius,
                                       float *kernel, int kernel_sz,
                                       int num_rows, int num_cols)
{
}
                                       

__global__ void filter_bleed_1_kernel(unsigned char *img_out, unsigned char *img_in,
                                      int radius, int kernel_sz,
                                      int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    unsigned char val_a = img_in[tx + ty * num_cols];
    int val_count = 0; 
    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + x;
            int sy = ty + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            unsigned char val_s = img_in[sx + sy * num_cols];
            if (val_s > 0)
                val_count = val_count + 1;
        }
    }

    if (val_count > (kernel_sz - 1) * 0.30)
        img_out[tx + ty * num_cols] = 1;
    else
        img_out[tx + ty * num_cols] = val_a;
}

void d_filter_bleed_1(unsigned char *d_img_out, unsigned char *d_img_in,
                    int radius,
                    int num_rows, int num_cols)
{
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);
    
    filter_bleed_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in, radius, kernel_sz, num_rows, num_cols);
    hipDeviceSynchronize();
}

void filter_bleed_1(unsigned char *img_out, unsigned char *img_in,
                    int radius,
                    int num_rows, int num_cols)
{
    cudaEventPair_t timer;

    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);

    unsigned char* d_img_in;
    unsigned char* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img_in, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_out, sizeof(unsigned char) * num_rows * num_cols));
    
    startCudaTimer(&timer);
    filter_bleed_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in, radius, kernel_sz, num_rows, num_cols);
    stopCudaTimer(&timer, "Bleed Filter (1 Component) Kernel");

    checkCudaError(hipMemcpy(img_out, d_img_out, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyDeviceToHost));

    hipFree(d_img_in);
    hipFree(d_img_out);
}

inline __device__ float gaussian1D(float x, float sigma)
{
    float variance = pow(sigma, 2);
    float power = pow(x, 2);
    float exponent = -power/(2*variance);
    return __expf(exponent) / sqrt(2 * PI * variance);
}

__global__ void filter_bilateral_1_kernel(float *img_out, float *img_in, float* kernel,
                                          int radius, float sigma_color, float sigma_spatial,
                                          int num_rows, int num_cols)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    float val_a = img_in[tx + ty * num_cols];

    int kernel_width = radius * 2 + 1;
    float norm = 0.0f;
    float res = 0.0f;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            int sx = tx + x;
            int sy = ty + y;

            if (sx < 0) sx = -sx;
            if (sy < 0) sy = -sy;
            if (sx > num_cols - 1) sx = num_cols - 1 - x;
            if (sy > num_rows - 1) sy = num_rows - 1 - y;

            float val_s = img_in[sx + sy * num_cols];

            float val_gspatial = kernel[(x + radius) + (y + radius) * kernel_width];
            float val_gcolor = gaussian1D(val_a - val_s, sigma_color);
            float weight = val_gspatial * val_gcolor;
            
            norm = norm + weight;
            res = res + (val_s * weight); 
        }
    }

    res /= norm;

    img_out[tx + ty * num_cols] = res;
}

__host__ __device__ float gaussian2D(float x, float y, float sigma)
{
    float variance = pow(sigma,2);
    float exponent = -(pow(x,2) + pow(y,2))/(2 * variance);
    return expf(exponent) / (2 * PI * variance);
}

void generateGaussianKernel(float* kernel, int radius, float sigma)
{
    int kernel_width = radius * 2 + 1;

    for (int y = -radius; y <= radius; ++y)
    {
        for (int x = -radius; x <= radius; ++x)
        {
            kernel[(x + radius) + (y + radius) * kernel_width] = gaussian2D(x, y, sigma);
        }
    }
}

void d_filter_bilateral_1(float *d_img,
                          int radius, float sigma_color, float sigma_spatial,
                          int num_rows, int num_cols)
{
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_out;
    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));
    
    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    filter_bilateral_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img, d_kernel, radius, sigma_color, sigma_spatial, num_rows, num_cols);
    hipDeviceSynchronize(); 

    checkCudaError(hipMemcpy(d_img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToDevice));
    
    hipFree(d_img_out);
    free(kernel);
    hipFree(d_kernel);
}

void filter_bilateral_1(float *img,
                        int radius, float sigma_color, float sigma_spatial,
                        int num_rows, int num_cols)
{
    cudaEventPair_t timer;
	
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    int kernel_sz = (2 * radius + 1) * (2 * radius + 1);
    float* kernel = (float*) malloc(sizeof(float) * kernel_sz);
    generateGaussianKernel(kernel, radius, sigma_spatial);
    
    // Device Memory Allocation & Copy
    float* d_img_in;
    float* d_img_out;

    checkCudaError(hipMalloc(&d_img_in, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_img_in, img, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_out, sizeof(float) * num_rows * num_cols));

    float* d_kernel;
    checkCudaError(hipMalloc(&d_kernel, sizeof(float) * kernel_sz));
    checkCudaError(hipMemcpy(d_kernel, kernel, sizeof(float) * kernel_sz, hipMemcpyHostToDevice));
    
    startCudaTimer(&timer);
    filter_bilateral_1_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in, d_kernel, radius, sigma_color, sigma_spatial, num_rows, num_cols);
    stopCudaTimer(&timer, "Bilateral Filter (1 Component) Kernel");
    
    checkCudaError(hipMemcpy(img, d_img_out, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));

    free(kernel);
    hipFree(d_kernel);
    hipFree(d_img_out);
    hipFree(d_img_in);
}


#endif
