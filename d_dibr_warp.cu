#include "hip/hip_runtime.h"
#ifndef D_DIBR_WARP_KERNEL 
#define D_DIBR_WARP_KERNEL
#include "d_dibr_warp.h"
#include "d_mux_common.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void dibr_forward_warp_kernel(unsigned char* img_out, unsigned char* holes, 
                                         unsigned char* img_in, float* disp,
                                         float shift, int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;
    
    int sd = (int) (disp[tx + ty * num_cols] * shift); 
    int sx = min(max(tx + sd, 0), num_cols - 1);
    //printf("%d ", sd);

    img_out[(sx + ty * num_cols) * elem_sz] = img_in[(tx + ty * num_cols) * elem_sz];
    img_out[(sx + ty * num_cols) * elem_sz + 1] = img_in[(tx + ty * num_cols) * elem_sz + 1];
    img_out[(sx + ty * num_cols) * elem_sz + 2] = img_in[(tx + ty * num_cols) * elem_sz + 2];
    
    holes[sx + ty * num_cols] = 1;
}


void dibr_dfm(unsigned char* img_out,
              unsigned char* img_in_l, unsigned char* img_in_r, float* disp_l, float* disp_r,
              float shift, int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////////////// 
    // MEMORY ALLOCATION //
    ///////////////////////
    float* d_disp_l, *d_disp_r;

    checkCudaError(hipMalloc(&d_disp_l, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_disp_r, sizeof(float) * num_rows * num_cols));

    checkCudaError(hipMemcpy(d_disp_l, disp_l, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_disp_r, disp_r, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    
    unsigned char* d_img_in_l, *d_img_in_r; 

    checkCudaError(hipMalloc(&d_img_in_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_img_in_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    checkCudaError(hipMemcpy(d_img_in_l, img_in_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_img_in_r, img_in_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    
    unsigned char* d_img_out_l, *d_img_out_r; 
    
    checkCudaError(hipMalloc(&d_img_out_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_img_out_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    
    checkCudaError(hipMemset(d_img_out_l, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemset(d_img_out_r, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    unsigned char* d_holes_l, *d_holes_r; 

    checkCudaError(hipMalloc(&d_holes_l, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_holes_r, sizeof(unsigned char) * num_rows * num_cols));
    
    checkCudaError(hipMemset(d_holes_l, 0, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMemset(d_holes_r, 0, sizeof(unsigned char) * num_rows * num_cols));
    
    /////////////////// 
    // LAUNCH KERNEL //
    ///////////////////
    
    startCudaTimer(&timer);
    dibr_forward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out_l, d_holes_l, d_img_in_l, d_disp_l, shift, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "DIBR Forward Map Kernel");
    
    startCudaTimer(&timer);
    dibr_forward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out_r, d_holes_r, d_img_in_r, d_disp_r, 1.0 - shift, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "DIBR Forward Map Kernel");
    
    startCudaTimer(&timer);
    mux_merge_AB_kernel<<<grid_sz, block_sz>>>(d_img_out_l, d_img_out_r, d_holes_l, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "Merge Kernel");
    
    ///////////////// 
    // MEMORY COPY //
    /////////////////

    checkCudaError(hipMemcpy(img_out, d_img_out_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyDeviceToHost));

    /////////////////// 
    // DE-ALLOCATION //
    ///////////////////

    hipFree(d_disp_l);
    hipFree(d_disp_r);
    hipFree(d_img_in_l);
    hipFree(d_img_in_r);
    hipFree(d_img_out_l);
    hipFree(d_img_out_r);
    hipFree(d_holes_l);
    hipFree(d_holes_r);
    hipFree(d_img_out_r);
}


#endif
