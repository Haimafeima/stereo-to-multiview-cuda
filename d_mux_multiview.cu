#include "hip/hip_runtime.h"
#ifndef D_MUX_MULTIVIEW_KERNEL
#define D_MUX_MULTIVIEW_KERNEL
#include "d_mux_multiview.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void mux_multiview_kernel(unsigned char** views, unsigned char* output, 
                                     int num_views, float angle,
									 int in_rows, int in_cols, int out_rows, int out_cols, int elem_sz)
{
    // Thread Id's
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= out_cols || ty >= out_rows)
        return;
    
    float PI = 3.141592653589793;
    
    // Compute Input Sampling Coordinates
    float x_samp = fmin(fmax(((float) tx / (float) out_cols) * (float) in_cols, 0), (float) (in_cols - 1));
    float y_samp = fmin(fmax(((float) ty / (float) out_rows) * (float) in_rows, 0), (float) (in_rows - 1));
    
    // Interlace Specific
    float x_interval = num_views;
    float y_interval = (float) num_views / tan(angle * PI / 180.0) / 3.0;
    float y_view = ty % ((int) round(y_interval)) + 1.0;
    y_view = y_view * x_interval / y_interval;
    int x_view = (tx * 3 + (int) y_view) % ((int) x_interval);
    int r_view = x_view;
    if (r_view < 0)
        r_view = r_view + num_views;
    int g_view = r_view + 1;
    int b_view = r_view + 2;
    if (g_view >= num_views)
        g_view = g_view - num_views;
    if (b_view >= num_views)
        b_view =  b_view - num_views;

    // Write to Output
    int b_out = (tx + ty * out_cols) * elem_sz;
    int g_out = b_out + 1;
    int r_out = g_out + 1;

    output[b_out] = alu_bilinear_interp(views[b_view], elem_sz, 0, x_samp, y_samp, in_cols, in_rows);
    output[g_out] = alu_bilinear_interp(views[g_view], elem_sz, 1, x_samp, y_samp, in_cols, in_rows);
    output[r_out] = alu_bilinear_interp(views[r_view], elem_sz, 2, x_samp, y_samp, in_cols, in_rows);
}

void d_mux_multiview( unsigned char **views, unsigned char* out_data, 
                      int num_views, float angle, 
				      int in_rows, int in_cols, int out_rows, int out_cols, int elem_sz)
{
    cudaEventPair_t timer;
	// Memory Allocation of Input
	unsigned char** d_views;
	checkCudaError(hipMalloc(&d_views, sizeof(unsigned char *) * num_views));
	
	unsigned char** h_views = (unsigned char**) malloc(sizeof(unsigned char**) * num_views);
	for (int v = 0; v < num_views; ++v)
	{
		checkCudaError(hipMalloc(&h_views[v], sizeof(unsigned char) * in_cols * in_rows * elem_sz));
		checkCudaError(hipMemcpy(h_views[v], views[v], sizeof(unsigned char) * in_cols * in_rows * elem_sz, hipMemcpyHostToDevice));
	}
	checkCudaError(hipMemcpy(d_views, h_views, sizeof(unsigned char *) * num_views, hipMemcpyHostToDevice));

	// Memory Allocation of Output
	unsigned char* d_out_data;
	checkCudaError(hipMalloc(&d_out_data, sizeof(unsigned char) * out_cols * out_rows * elem_sz));
    
	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (out_cols + bw - 1) / bw;
    size_t gh = (out_rows + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
 	
    // Launch Kernel
    startCudaTimer(&timer);
 	mux_multiview_kernel<<<grid_sz, block_sz>>>(d_views, d_out_data, angle, num_views, in_rows, in_cols, out_rows, out_cols, elem_sz);
    hipDeviceSynchronize();
    stopCudaTimer(&timer, "Multiview Interlace Kernel");

	// Copy Memory back to Host
	checkCudaError(hipMemcpy(out_data, d_out_data, sizeof(unsigned char) * out_cols * out_rows * elem_sz,hipMemcpyDeviceToHost));

	// De-allocation of Host & Device Memory
	for (int v = 0; v < num_views; ++v)
		hipFree(h_views[v]);
	hipFree(d_views);
	hipFree(d_out_data);
    free(h_views);
}

#endif
