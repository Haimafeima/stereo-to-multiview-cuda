#include "hip/hip_runtime.h"
#ifndef D_MUX_MULTIVIEW_KERNEL
#define D_MUX_MULTIVIEW_KERNEL
#include "d_mux_multiview.h"
#include "cuda_utils.h"
#include "d_alu.h"
#include <math.h>

__global__ void mux_multiview_kernel(unsigned char** views, unsigned char* output, 
                                     int num_views, float angle,
									 int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
    // Thread Id's
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= num_cols_out || ty >= num_rows_out)
        return;
    
    float PI = 3.141592653589793;
    
    // Compute Input Sampling Coordinates
    float x_samp = fmin(fmax(((float) tx / (float) num_cols_out) * (float) num_cols_in, 0), (float) (num_cols_in - 1));
    float y_samp = fmin(fmax(((float) ty / (float) num_rows_out) * (float) num_rows_in, 0), (float) (num_rows_in - 1));
    
    // Interlace Specific
    float x_interval = num_views;
    float y_interval = (float) num_views / tan(angle * PI / 180.0) / 3.0;
    float y_view = ty % ((int) round(y_interval)) + 1.0;
    y_view = y_view * x_interval / y_interval;
    int x_view = (tx * 3 + (int) y_view) % ((int) x_interval);
    int r_view = x_view;
    if (r_view < 0)
        r_view = r_view + num_views;
    int g_view = r_view + 1;
    int b_view = r_view + 2;
    if (g_view >= num_views)
        g_view = g_view - num_views;
    if (b_view >= num_views)
        b_view =  b_view - num_views;
    
    // Write to Output
    int b_out = (tx + ty * num_cols_out) * elem_sz;
    int g_out = b_out + 1;
    int r_out = g_out + 1;

    output[b_out] = alu_bilinear_interp(views[b_view], elem_sz, 0, x_samp, y_samp, num_cols_in, num_rows_in);
    output[g_out] = alu_bilinear_interp(views[g_view], elem_sz, 1, x_samp, y_samp, num_cols_in, num_rows_in);
    output[r_out] = alu_bilinear_interp(views[r_view], elem_sz, 2, x_samp, y_samp, num_cols_in, num_rows_in);
}

void d_mux_multiview( unsigned char **d_views, unsigned char* d_out_data, 
                      int num_views, float angle, 
				      int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols_out + bw - 1) / bw;
    size_t gh = (num_rows_out + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
 	
    // Launch Kernel
 	mux_multiview_kernel<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, angle, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
    hipDeviceSynchronize(); 
}


void mux_multiview( unsigned char **views, unsigned char* out_data, 
                      int num_views, float angle, 
				      int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
    cudaEventPair_t timer;
	// Memory Allocation of Input
	unsigned char** d_views;
	checkCudaError(hipMalloc(&d_views, sizeof(unsigned char *) * num_views));
	
	unsigned char** h_views = (unsigned char**) malloc(sizeof(unsigned char*) * num_views);
	for (int v = 0; v < num_views; ++v)
	{
		checkCudaError(hipMalloc(&h_views[v], sizeof(unsigned char) * num_cols_in * num_rows_in * elem_sz));
		checkCudaError(hipMemcpy(h_views[v], views[v], sizeof(unsigned char) * num_cols_in * num_rows_in * elem_sz, hipMemcpyHostToDevice));
	}
	checkCudaError(hipMemcpy(d_views, h_views, sizeof(unsigned char *) * num_views, hipMemcpyHostToDevice));

	// Memory Allocation of Output
	unsigned char* d_out_data;
	checkCudaError(hipMalloc(&d_out_data, sizeof(unsigned char) * num_cols_out * num_rows_out * elem_sz));
    
	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    
    size_t gw = (num_cols_out + bw - 1) / bw;
    size_t gh = (num_rows_out + bh - 1) / bh;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
 	
    // Launch Kernel
    startCudaTimer(&timer);
 	mux_multiview_kernel<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, angle, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
    stopCudaTimer(&timer, "Multiview Interlace Kernel");

	// Copy Memory back to Host
	checkCudaError(hipMemcpy(out_data, d_out_data, sizeof(unsigned char) * num_cols_out * num_rows_out * elem_sz,hipMemcpyDeviceToHost));

	// De-allocation of Host & Device Memory
	for (int v = 0; v < num_views; ++v)
		hipFree(h_views[v]);
	hipFree(d_views);
	hipFree(d_out_data);
    free(h_views);
}

#endif
