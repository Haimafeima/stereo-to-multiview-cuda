#include "hip/hip_runtime.h"
#ifndef D_MUX_MULTIVIEW_KERNEL
#define D_MUX_MULTIVIEW_KERNEL
#include "d_mux_multiview.h"
#include "cuda_utils.h"
#include "d_alu.h"
#include <math.h>

#define PI 3.1415926535f

inline __device__ unsigned char fast_bilinear_interp(unsigned char* data, int elem_sz, int elem_offset, float coord_x, float coord_y, int width, int height) 
{
    int coord_00_x = floor(coord_x);
    int coord_00_y = floor(coord_y);
    
    int coord_01_x = min(coord_00_x + 1, width - 1);
    int coord_01_y = coord_00_y;

    int coord_10_x = coord_00_x;
    int coord_10_y = min(coord_00_y + 1, height - 1);
    
    int coord_11_x = min(coord_00_x + 1, width - 1);
    int coord_11_y = min(coord_00_y + 1, height - 1);

    float weight_x = coord_x - (float) coord_00_x;
    float weight_y = coord_y - (float) coord_00_y;

    unsigned char val_00 = data[(coord_00_x + coord_00_y * width) * elem_sz + elem_offset];
    unsigned char val_01 = data[(coord_01_x + coord_01_y * width) * elem_sz + elem_offset];
    unsigned char val_10 = data[(coord_10_x + coord_10_y * width) * elem_sz + elem_offset];
    unsigned char val_11 = data[(coord_11_x + coord_11_y * width) * elem_sz + elem_offset];

    float top = (float) val_00 * (1.0f - weight_x) + (float) val_01 * weight_x;
    float bot = (float) val_10 * (1.0f - weight_x) + (float) val_11 * weight_x;
    
    return (unsigned char) (top * (1.0f - weight_y) + bot * weight_y);
}

__global__ void mux_multiview_kernel_2(unsigned char** views, unsigned char* output, 
                                       int num_views, float y_interval, float inv_y_interval,
									   int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
    // Thread Id's
    int gx = num_views * threadIdx.x + blockIdx.x * blockDim.x;
    int gy = threadIdx.y + blockIdx.y * blockDim.y;

    if (gx >= num_cols_out || gy >= num_rows_out)
        return;

    // Compute Input Sampling Coordinates
    for (int v = 0; v < num_views; ++v)
    {
        int tx = gx + v;
        if (tx > num_cols_out) 
            return;
        int ty = gy;

        float x_samp = fmin(fmax(((float) tx / (float) num_cols_out) * (float) num_cols_in, 0), (float) (num_cols_in - 1));
        float y_samp = fmin(fmax(((float) ty / (float) num_rows_out) * (float) num_rows_in, 0), (float) (num_rows_in - 1));
        
        // Interlace Specific
        float x_interval = num_views;
        float y_view = ty % ((int) round(y_interval)) + 1.0f;
        y_view = y_view * x_interval * inv_y_interval;
        int x_view = (tx * 3 + (int) y_view) % ((int) x_interval);
        int r_view = x_view;
        if (r_view < 0)
            r_view = r_view + num_views;
        int g_view = r_view + 1;
        int b_view = r_view + 2;
        if (g_view >= num_views)
            g_view = g_view - num_views;
        if (b_view >= num_views)
            b_view =  b_view - num_views;
        
        // Write to Output
        int b_out = (tx + ty * num_cols_out) * elem_sz;
        int g_out = b_out + 1;
        int r_out = g_out + 1;

        output[b_out] = fast_bilinear_interp(views[b_view], elem_sz, 0, x_samp, y_samp, num_cols_in, num_rows_in);
        output[g_out] = fast_bilinear_interp(views[g_view], elem_sz, 1, x_samp, y_samp, num_cols_in, num_rows_in);
        output[r_out] = fast_bilinear_interp(views[r_view], elem_sz, 2, x_samp, y_samp, num_cols_in, num_rows_in);
    }
}

__global__ void mux_multiview_kernel(unsigned char** views, unsigned char* output, 
                                     int num_views, float y_interval,
									 int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
    // Thread Id's
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    if (tx >= num_cols_out || ty >= num_rows_out)
        return;
    
    // Compute Input Sampling Coordinates
    float x_samp = fmin(fmax(((float) tx / (float) num_cols_out) * (float) num_cols_in, 0), (float) (num_cols_in - 1));
    float y_samp = fmin(fmax(((float) ty / (float) num_rows_out) * (float) num_rows_in, 0), (float) (num_rows_in - 1));
    
    // Interlace Specific
    float x_interval = num_views;
    float y_view = ty % ((int) round(y_interval)) + 1.0;
    y_view = y_view * x_interval / y_interval;
    int x_view = (tx * 3 + (int) y_view) % ((int) x_interval);
    int r_view = x_view;
    if (r_view < 0)
        r_view = r_view + num_views;
    int g_view = r_view + 1;
    int b_view = r_view + 2;
    if (g_view >= num_views)
        g_view = g_view - num_views;
    if (b_view >= num_views)
        b_view =  b_view - num_views;
    
    // Write to Output
    int b_out = (tx + ty * num_cols_out) * elem_sz;
    int g_out = b_out + 1;
    int r_out = g_out + 1;

    output[b_out] = fast_bilinear_interp(views[b_view], elem_sz, 0, x_samp, y_samp, num_cols_in, num_rows_in);
    output[g_out] = fast_bilinear_interp(views[g_view], elem_sz, 1, x_samp, y_samp, num_cols_in, num_rows_in);
    output[r_out] = fast_bilinear_interp(views[r_view], elem_sz, 2, x_samp, y_samp, num_cols_in, num_rows_in);
}

void d_mux_multiview( unsigned char **d_views, unsigned char* d_out_data, 
                      int num_views, float angle, 
				      int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols_out + bw - 1) / bw;
    size_t gh = (num_rows_out + bh - 1) / bh;
    
    int kernel_num = 0;
    bw = num_cols_out / num_views;
    bh = 1;
    gw = (num_cols_out + bw - 1) / bw / num_views;
    gh = (num_rows_out + bh - 1) / bh;
    kernel_num = 1;
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    float y_interval = (float) num_views / tan(angle * PI / 180.0) / (float) elem_sz;
    
    if (kernel_num == 0)
        mux_multiview_kernel<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, y_interval, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
    else if (kernel_num == 1) 
        mux_multiview_kernel_2<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, y_interval, 1.0f/y_interval, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
    hipDeviceSynchronize();
}


void mux_multiview( unsigned char **views, unsigned char* out_data, 
                      int num_views, float angle, 
				      int num_rows_in, int num_cols_in, int num_rows_out, int num_cols_out, int elem_sz)
{
    cudaEventPair_t timer;
	// Memory Allocation of Input
	unsigned char** d_views;
	checkCudaError(hipMalloc(&d_views, sizeof(unsigned char *) * num_views));
	
	unsigned char** h_views = (unsigned char**) malloc(sizeof(unsigned char*) * num_views);
	for (int v = 0; v < num_views; ++v)
	{
		checkCudaError(hipMalloc(&h_views[v], sizeof(unsigned char) * num_cols_in * num_rows_in * elem_sz));
		checkCudaError(hipMemcpy(h_views[v], views[v], sizeof(unsigned char) * num_cols_in * num_rows_in * elem_sz, hipMemcpyHostToDevice));
	}
	checkCudaError(hipMemcpy(d_views, h_views, sizeof(unsigned char *) * num_views, hipMemcpyHostToDevice));

	// Memory Allocation of Output
	unsigned char* d_out_data;
	checkCudaError(hipMalloc(&d_out_data, sizeof(unsigned char) * num_cols_out * num_rows_out * elem_sz));
    
	// Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols_out + bw - 1) / bw;
    size_t gh = (num_rows_out + bh - 1) / bh;
    
    int kernel_num = 0;
    if (num_rows_out % num_views == 0)
    {
        bw = num_cols_out / num_views;
        bh = 1;
        gw = (num_cols_out + bw - 1) / bw / num_views;
        gh = (num_rows_out + bh - 1) / bh;
        kernel_num = 1;
    }
    
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    float y_interval = (float) num_views / tan(angle * PI / 180.0) / (float) elem_sz;

    if (kernel_num == 0)
    {
        startCudaTimer(&timer);
        mux_multiview_kernel<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, y_interval, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
        stopCudaTimer(&timer, "Multiview Interlace Kernel");
    }
    else if (kernel_num == 1) 
    {
        startCudaTimer(&timer);
        mux_multiview_kernel_2<<<grid_sz, block_sz>>>(d_views, d_out_data, num_views, y_interval, 1.0f/y_interval, num_rows_in, num_cols_in, num_rows_out, num_cols_out, elem_sz);
        stopCudaTimer(&timer, "Multiview Interlace Kernel #2");
    }

	// Copy Memory back to Host
	checkCudaError(hipMemcpy(out_data, d_out_data, sizeof(unsigned char) * num_cols_out * num_rows_out * elem_sz,hipMemcpyDeviceToHost));

	// De-allocation of Host & Device Memory
	for (int v = 0; v < num_views; ++v)
		hipFree(h_views[v]);
	hipFree(d_views);
	hipFree(d_out_data);
    free(h_views);
}

#endif
