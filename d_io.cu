#ifndef D_IO_KERNEL 
#define D_IO_KERNEL
#include "d_io.h"

void adcensus_stm(unsigned char *img_sbs, float *disp_l, float *disp_r,
                  unsigned char* interlaced,
                  int num_rows, int num_cols_sbs, int num_cols, 
                  int num_rows_out, int num_cols_out, int elem_sz,
                  int num_views, int angle,
                  int num_disp, int zero_disp,
                  float ad_coeff, float census_coeff,
                  float ucd, float lcd, int usd, int lsd)
{
    ///////////
    // SIZES //
    ///////////

    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    size_t img_sz = num_rows * num_cols;
    size_t imgelem_sz = img_sz * elem_sz;
    size_t cost_sz = img_sz * num_disp;
    
    size_t gw_sbs = (num_cols_sbs + bw - 1) / bw;
    const dim3 grid_sz_sbs(gw_sbs, gh, 1);
    
    ///////////////////////
    // MEMORY ALLOCATION //
    ///////////////////////

    unsigned char* d_img_sbs;    
    unsigned char* d_img_l;
    unsigned char* d_img_r;

    checkCudaError(hipMalloc(&d_img_sbs, sizeof(unsigned char) * num_rows * num_cols_sbs * elem_sz));
    checkCudaError(hipMemcpy(d_img_sbs, img_sbs, sizeof(unsigned char) * num_rows * num_cols_sbs * elem_sz, hipMemcpyHostToDevice)); 

    checkCudaError(hipMalloc(&d_img_l, sizeof(unsigned char) * imgelem_sz));
    checkCudaError(hipMalloc(&d_img_r, sizeof(unsigned char) * imgelem_sz));
    
    ///////////////////////////
    // SIDE BY SIDE SPLITTER //
    ///////////////////////////

    demux_sbs<<<grid_sz_sbs, block_sz>>>(d_img_l, d_img_r, d_img_sbs, num_rows, num_cols_sbs, num_cols, elem_sz);
    hipDeviceSynchronize();
    
    hipFree(d_img_sbs);

    /////////////////////////
    // COST INITIALIZATION //
    /////////////////////////
    
    float** d_adcensus_cost_l;
    float** d_adcensus_cost_r;
    
    checkCudaError(hipMalloc(&d_adcensus_cost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_adcensus_cost_r, sizeof(float*) * num_disp));
   
    float** h_adcensus_cost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_adcensus_cost_r = (float**) malloc(sizeof(float*) * num_disp);

    float* d_adcensus_cost_memory;
    checkCudaError(hipMalloc(&d_adcensus_cost_memory, sizeof(float) * cost_sz * 2));

    d_ci_adcensus(d_img_l, d_img_r, d_adcensus_cost_l, d_adcensus_cost_r, 
                  h_adcensus_cost_l, h_adcensus_cost_r, d_adcensus_cost_memory,
                  ad_coeff, census_coeff, 
                  num_disp, zero_disp, num_rows, num_cols, elem_sz);
    
    //////////////////////
    // COST AGGRAGATION //
    //////////////////////

    float** d_acost_l, **d_acost_r;

    checkCudaError(hipMalloc(&d_acost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_acost_r, sizeof(float*) * num_disp));

    float** h_acost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_acost_r = (float**) malloc(sizeof(float*) * num_disp);

    float* d_acost_memory;
    checkCudaError(hipMalloc(&d_acost_memory, sizeof(float) * cost_sz * 2));
    
    d_ca_cross(d_img_l, d_adcensus_cost_l, d_acost_l, h_acost_l, d_acost_memory, ucd, lcd, usd, lsd, num_disp, num_rows, num_cols, elem_sz);
    
    d_ca_cross(d_img_r, d_adcensus_cost_r, d_acost_r, h_acost_r, d_acost_memory + cost_sz, ucd, lcd, usd, lsd, num_disp, num_rows, num_cols, elem_sz);

    hipFree(d_acost_l);
    hipFree(d_acost_r);
    hipFree(d_acost_memory);
    free(h_acost_l); 
    free(h_acost_r); 

    ///////////////////////////
    // DISPARITY COMPUTATION //
    ///////////////////////////
    
    float* d_disp_l, *d_disp_r;
    
    checkCudaError(hipMalloc(&d_disp_l, sizeof(float) * img_sz));
    checkCudaError(hipMalloc(&d_disp_r, sizeof(float) * img_sz));
	
	d_dc_wta(d_adcensus_cost_l, d_disp_l, num_disp, zero_disp, num_rows, num_cols);
    d_dc_wta(d_adcensus_cost_r, d_disp_r, num_disp, zero_disp, num_rows, num_cols);
	
	hipFree(d_adcensus_cost_l);
    hipFree(d_adcensus_cost_r);
    hipFree(d_adcensus_cost_memory);
    free(h_adcensus_cost_l); 
    free(h_adcensus_cost_r); 
    
    d_filter_bilateral_1(d_disp_l, 7, 5, 10, num_rows, num_cols, num_disp);
    d_filter_bilateral_1(d_disp_r, 7, 5, 10, num_rows, num_cols, num_disp);

    checkCudaError(hipMemcpy(disp_l, d_disp_l, sizeof(float) * img_sz, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(disp_r, d_disp_r, sizeof(float) * img_sz, hipMemcpyDeviceToHost));
     
    
    //////////
    // DIBR //
    //////////

    unsigned char *d_occl_l, *d_occl_r;
    
    checkCudaError(hipMalloc(&d_occl_l, sizeof(unsigned char) * img_sz));
    checkCudaError(hipMalloc(&d_occl_r, sizeof(unsigned char) * img_sz));
    
    d_dibr_occl(d_occl_l, d_occl_r, d_disp_l, d_disp_r, num_rows, num_cols);

    d_filter_bleed_1(d_occl_l, 1, num_rows, num_cols);    
    d_filter_bleed_1(d_occl_r, 1, num_rows, num_cols);

    float *d_mask_l, *d_mask_r;

    checkCudaError(hipMalloc(&d_mask_l, sizeof(float) * img_sz));
    checkCudaError(hipMalloc(&d_mask_r, sizeof(float) * img_sz));
    
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_l, d_occl_l, num_rows, num_cols);
    dibr_occl_to_mask_kernel<<<grid_sz, block_sz>>>(d_mask_r, d_occl_r, num_rows, num_cols);
    
    unsigned char* d_views_memory;
    checkCudaError(hipMalloc(&d_views_memory, sizeof(unsigned char) * imgelem_sz * num_views));

    unsigned char** h_views = (unsigned char**) malloc(sizeof(unsigned char*) * num_views);
    h_views[0] = d_img_r;
    h_views[num_views - 1] = d_img_l;
    for (int v = 1; v < num_views - 1; ++v)
        h_views[v] = d_views_memory + (v * imgelem_sz);
    
    for (int v = 1; v < num_views - 1; ++v)
    {   
        float shift = 1.0 - ((1.0 * (float) v) / ((float) num_views - 1.0));
        d_dibr_dbm(h_views[v], d_img_l, d_img_r, d_disp_l, d_disp_r, d_occl_l, d_occl_r, d_mask_l, d_mask_r, shift, num_rows, num_cols, elem_sz);
    }

    /////////
    // MUX //
    /////////
    unsigned char** d_views;
    checkCudaError(hipMalloc(&d_views, sizeof(unsigned char*) * num_views));
    checkCudaError(hipMemcpy(d_views, h_views, sizeof(unsigned char*) * num_views, hipMemcpyHostToDevice));

    unsigned char* d_interlaced;
    checkCudaError(hipMalloc(&d_interlaced, sizeof(unsigned char) * num_rows_out * num_cols_out * elem_sz));

    d_mux_multiview(d_views, d_interlaced, num_views, angle, num_rows, num_cols, num_rows_out, num_cols_out, elem_sz);

    checkCudaError(hipMemcpy(interlaced, d_interlaced, sizeof(unsigned char) * num_rows_out * num_cols_out * elem_sz, hipMemcpyDeviceToHost));
    
    ///////////////////
    // DE-ALLOCATION //
    ///////////////////
    
    hipFree(d_img_l);
    hipFree(d_img_r);
    
    hipFree(d_disp_l);
    hipFree(d_disp_r);
    hipFree(d_occl_l);
    hipFree(d_occl_r);
    hipFree(d_mask_l);
    hipFree(d_mask_r);
    
    hipFree(d_views_memory);
    hipFree(d_views);

    hipFree(d_interlaced);
    
    free(h_views);
}


#endif
