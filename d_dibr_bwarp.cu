#include "hip/hip_runtime.h"
#ifndef D_DIBR_BWARP_KERNEL 
#define D_DIBR_BWARP_KERNEL
#include "d_dibr_bwarp.h"
#include "d_op.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void dibr_backward_warp_kernel(unsigned char* img_out, unsigned char* img_in,
                                          float* mask, float *disp,
                                          float shift, int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    float val_mask = mask[tx + ty * num_cols];
    float sd = (disp[tx + ty * num_cols] * shift); 
    int sx = min(max((float) tx + sd, 0.0f), (float)(num_cols - 1));

    img_out[(tx + ty * num_cols) * elem_sz] = (unsigned char) ((float) alu_bilinear_interp(img_in, elem_sz, 0, sx, (float) ty, num_cols, num_rows) * val_mask);
    img_out[(tx + ty * num_cols) * elem_sz + 1] = (unsigned char) ((float) alu_bilinear_interp(img_in, elem_sz, 1, sx, (float) ty, num_cols, num_rows) * val_mask);
    img_out[(tx + ty * num_cols) * elem_sz + 2] = (unsigned char) ((float) alu_bilinear_interp(img_in, elem_sz, 2, sx, (float) ty, num_cols, num_rows) * val_mask);
}

void d_dibr_dbm(unsigned char* d_img_out,
                unsigned char* d_img_in_l, unsigned char* d_img_in_r, 
                float* d_disp_l, float* d_disp_r,
                unsigned char *d_occl_l, unsigned char *d_occl_r,
                float* d_mask_l, float* d_mask_r,
                float shift, int num_rows, int num_cols, int elem_sz)
{
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    //////////// 
    // KERNEL //
    ////////////
    
    float * d_tempmask_r;
    checkCudaError(hipMalloc(&d_tempmask_r, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMemcpy(d_tempmask_r, d_mask_r, sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToDevice));

    unsigned char* d_img_out_r; 
    checkCudaError(hipMalloc(&d_img_out_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    
    checkCudaError(hipMemset(d_img_out, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemset(d_img_out_r, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    dibr_backward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_in_l, d_mask_r, d_disp_r, -shift, num_rows, num_cols, elem_sz);   
    dibr_backward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out_r, d_img_in_r, d_mask_l, d_disp_l, 1.0 - shift, num_rows, num_cols, elem_sz);
    hipDeviceSynchronize();
    
    op_invertnormf_kernel<<<grid_sz, block_sz>>>(d_tempmask_r, num_rows, num_cols);
    hipDeviceSynchronize(); 
    
    d_filter_gaussian_1(d_tempmask_r, 10, 15, num_rows, num_cols);

    mux_merge_AB_kernel<<<grid_sz, block_sz>>>(d_img_out, d_img_out_r, d_tempmask_r, num_rows, num_cols, elem_sz);  
    hipDeviceSynchronize(); 

    hipFree(d_img_out_r);
    hipFree(d_tempmask_r);
}


void dibr_dbm(unsigned char* img_out,
              unsigned char* img_in_l, unsigned char* img_in_r, 
              float* disp_l, float* disp_r,
              unsigned char *occl_l, unsigned char *occl_r,
              float *mask_l, float *mask_r,
              float shift, int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    /////////////////////// 
    // DEVICE PARAMETERS //
    ///////////////////////
    
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);
    
    /////////////// 
    // OCCLUSION //
    ///////////////
    unsigned char* d_occl_l, *d_occl_r; 

    checkCudaError(hipMalloc(&d_occl_l, sizeof(unsigned char) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_occl_r, sizeof(unsigned char) * num_rows * num_cols));
    
    checkCudaError(hipMemcpy(d_occl_l, occl_l, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_occl_r, occl_r, sizeof(unsigned char) * num_rows * num_cols, hipMemcpyHostToDevice));
    
    float* d_mask_l, *d_mask_r; 

    checkCudaError(hipMalloc(&d_mask_l, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_mask_r, sizeof(float) * num_rows * num_cols));
    
    checkCudaError(hipMemcpy(d_mask_l, mask_l, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_mask_r, mask_r, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    
    /////////////////////// 
    // MEMORY ALLOCATION //
    ///////////////////////
    float* d_disp_l, *d_disp_r;

    checkCudaError(hipMalloc(&d_disp_l, sizeof(float) * num_rows * num_cols));
    checkCudaError(hipMalloc(&d_disp_r, sizeof(float) * num_rows * num_cols));

    checkCudaError(hipMemcpy(d_disp_l, disp_l, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_disp_r, disp_r, sizeof(float) * num_rows * num_cols, hipMemcpyHostToDevice));
    
    unsigned char* d_img_in_l, *d_img_in_r; 

    checkCudaError(hipMalloc(&d_img_in_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_img_in_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    checkCudaError(hipMemcpy(d_img_in_l, img_in_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_img_in_r, img_in_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    
    unsigned char* d_img_out_l, *d_img_out_r; 
    
    checkCudaError(hipMalloc(&d_img_out_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_img_out_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    
    checkCudaError(hipMemset(d_img_out_l, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemset(d_img_out_r, 0, sizeof(unsigned char) * num_rows * num_cols * elem_sz));

    startCudaTimer(&timer);
    dibr_backward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out_l, d_img_in_l, d_mask_r, d_disp_r, -shift, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "DIBR Backward Map Kernel");
    
    startCudaTimer(&timer);
    dibr_backward_warp_kernel<<<grid_sz, block_sz>>>(d_img_out_r, d_img_in_r, d_mask_l, d_disp_l, 1.0f - shift, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "DIBR Backward Map Kernel");
    
    startCudaTimer(&timer);
    op_invertnormf_kernel<<<grid_sz, block_sz>>>(d_mask_r, num_rows, num_cols);
    stopCudaTimer(&timer, "OP Invert Normalized Float Map Kernel");
    
    d_filter_gaussian_1(d_mask_r, 10, 15, num_rows, num_cols);
    
    startCudaTimer(&timer);
    mux_merge_AB_kernel<<<grid_sz, block_sz>>>(d_img_out_l, d_img_out_r, d_mask_r, num_rows, num_cols, elem_sz);  
    stopCudaTimer(&timer, "Merge Kernel");
    
    //startCudaTimer(&timer);
    //op_invertnormf_kernel<<<grid_sz, block_sz>>>(d_mask_r, num_rows, num_cols);
    //stopCudaTimer(&timer, "OP Invert Normalized Float Map Kernel");
    ///////////////// 
    // MEMORY COPY //
    /////////////////

    checkCudaError(hipMemcpy(img_out, d_img_out_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyDeviceToHost));

    /////////////////// 
    // DE-ALLOCATION //
    ///////////////////

    hipFree(d_disp_l);
    hipFree(d_disp_r);
    hipFree(d_img_in_l);
    hipFree(d_img_in_r);
    hipFree(d_img_out_l);
    hipFree(d_img_out_r);
    hipFree(d_occl_l);
    hipFree(d_occl_r);
    hipFree(d_mask_l);
    hipFree(d_mask_r);
}

#endif

