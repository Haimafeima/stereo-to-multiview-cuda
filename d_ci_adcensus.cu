#include "hip/hip_runtime.h"
#ifndef D_CI_ADCENSUS_KERNEL 
#define D_CI_ADCENSUS_KERNEL
#include "d_ci_adcensus.h"
#include "d_ci_ad.h"
#include "d_ci_census.h"
#include "cuda_utils.h"
#include <math.h>

__global__ void ci_adcensus_kernel(float** ad_cost_l, float** ad_cost_r, float** census_cost_l, float** census_cost_r,
                                   float** adcensus_cost_l, float** adcensus_cost_r,
                                   float inv_ad_coeff, float inv_census_coeff, int num_disp, int zero_disp, 
                                   int num_rows, int num_cols, int elem_sz)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    
    if ((tx > num_cols - 1) || (ty > num_rows - 1))
        return;

    for (int d = 0; d < num_disp; ++d)
    {
       float ad_comp_l = 1.0 - exp(-ad_cost_l[d][tx + ty * num_cols]*inv_ad_coeff);
       float ad_comp_r = 1.0 - exp(-ad_cost_r[d][tx + ty * num_cols]*inv_ad_coeff);

       float census_comp_l = 1.0 - exp(-census_cost_l[d][tx + ty * num_cols]*inv_census_coeff);
       float census_comp_r = 1.0 - exp(-census_cost_r[d][tx + ty * num_cols]*inv_census_coeff);

       adcensus_cost_l[d][tx + ty * num_cols] = ad_comp_l + census_comp_l;
       adcensus_cost_r[d][tx + ty * num_cols] = ad_comp_r + census_comp_r;
    }
}


void ci_adcensus(unsigned char* img_l, unsigned char* img_r, float** cost_l, float** cost_r, 
                 float ad_coeff, float census_coeff, int num_disp, int zero_disp, 
                 int num_rows, int num_cols, int elem_sz)
{
    cudaEventPair_t timer;
    
    //////////// 
    // COMMON //
    ////////////
    
    // Device Image Memory Allocation
    unsigned char* d_img_l;
    unsigned char* d_img_r;

    checkCudaError(hipMalloc(&d_img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img_l, img_l, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));

    checkCudaError(hipMalloc(&d_img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMemcpy(d_img_r, img_r, sizeof(unsigned char) * num_rows * num_cols * elem_sz, hipMemcpyHostToDevice));
    
    // Setup Block & Grid Size
    size_t bw = 32;
    size_t bh = 32;
    size_t gw = (num_cols + bw - 1) / bw;
    size_t gh = (num_rows + bh - 1) / bh;
    const dim3 block_sz(bw, bh, 1);
    const dim3 grid_sz(gw, gh, 1);

    ////////
    // AD //
    ////////

    // Device Memory Allocation & Copy
    float** d_ad_cost_l;
    float** d_ad_cost_r;
    
    checkCudaError(hipMalloc(&d_ad_cost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_ad_cost_r, sizeof(float*) * num_disp));
    
    float** h_ad_cost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_ad_cost_r = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_ad_cost_l[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMalloc(&h_ad_cost_r[d], sizeof(float) * num_rows * num_cols));
    }
    
    checkCudaError(hipMemcpy(d_ad_cost_l, h_ad_cost_l, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_ad_cost_r, h_ad_cost_r, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    // Launch Kernel
    startCudaTimer(&timer);
    ci_ad_kernel<<<grid_sz, block_sz>>>(d_img_l, d_img_r, d_ad_cost_l, d_ad_cost_r, num_disp, zero_disp, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Absolute Difference Kernel");

    ////////////
    // CENSUS //
    ////////////

    // Device Memory Allocation & Copy
    unsigned long long* d_census_l;
    unsigned long long* d_census_r;

    checkCudaError(hipMalloc(&d_census_l, sizeof(unsigned long long) * num_rows * num_cols * elem_sz));
    checkCudaError(hipMalloc(&d_census_r, sizeof(unsigned long long) * num_rows * num_cols * elem_sz));

    // Launch Census Transform Kernel
    startCudaTimer(&timer);
    tx_census_9x7_kernel<<<grid_sz, block_sz>>>(d_img_l, d_census_l, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Census Transform Kernel");
    
    startCudaTimer(&timer);
    tx_census_9x7_kernel<<<grid_sz, block_sz>>>(d_img_r, d_census_r, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Census Transform Kernel");
    
    // Cost Initialization Device Cost Memory
    float** d_census_cost_l;
    float** d_census_cost_r;
    
    checkCudaError(hipMalloc(&d_census_cost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_census_cost_r, sizeof(float*) * num_disp));
    
    float** h_census_cost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_census_cost_r = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_census_cost_l[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMalloc(&h_census_cost_r[d], sizeof(float) * num_rows * num_cols));
    }
    
    checkCudaError(hipMemcpy(d_census_cost_l, h_census_cost_l, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_census_cost_r, h_census_cost_r, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    // Launch Kernel
    startCudaTimer(&timer);
    ci_census_kernel<<<grid_sz, block_sz>>>(d_census_r, d_census_r, d_census_cost_l, d_census_cost_r, num_disp, zero_disp, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Census Cost Kernel");
    
    /////////////////
    // AD + CENSUS //
    /////////////////
    
    float** d_adcensus_cost_l;
    float** d_adcensus_cost_r;
    
    checkCudaError(hipMalloc(&d_adcensus_cost_l, sizeof(float*) * num_disp));
    checkCudaError(hipMalloc(&d_adcensus_cost_r, sizeof(float*) * num_disp));
    
    float** h_adcensus_cost_l = (float**) malloc(sizeof(float*) * num_disp);
    float** h_adcensus_cost_r = (float**) malloc(sizeof(float*) * num_disp);
    
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMalloc(&h_adcensus_cost_l[d], sizeof(float) * num_rows * num_cols));
        checkCudaError(hipMalloc(&h_adcensus_cost_r[d], sizeof(float) * num_rows * num_cols));
    }
    
    checkCudaError(hipMemcpy(d_adcensus_cost_l, h_adcensus_cost_l, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_adcensus_cost_r, h_adcensus_cost_r, sizeof(float*) * num_disp, hipMemcpyHostToDevice));
    
    // Launch Kernel
    startCudaTimer(&timer);
    ci_adcensus_kernel<<<grid_sz, block_sz>>>(d_ad_cost_l, d_ad_cost_r, d_census_cost_l, d_census_cost_r, d_adcensus_cost_l, d_adcensus_cost_r, 1.0/ad_coeff, 1.0/census_coeff, num_disp, zero_disp, num_rows, num_cols, elem_sz);
    stopCudaTimer(&timer, "Ad + Census Cost Kernel");

    // Copy Memory Device -> Host
    for (int d = 0; d < num_disp; ++d)
    {
        checkCudaError(hipMemcpy(cost_l[d], h_adcensus_cost_l[d], sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
        checkCudaError(hipMemcpy(cost_r[d], h_adcensus_cost_r[d], sizeof(float) * num_rows * num_cols, hipMemcpyDeviceToHost));
    }

    /////////
    // END //
    /////////
    
    // Device De-allocation
    hipFree(d_img_l);
    hipFree(d_img_r);
    hipFree(d_census_l);
    hipFree(d_census_r);
    hipFree(d_ad_cost_l);
    hipFree(d_ad_cost_r);
    hipFree(d_census_cost_l);
    hipFree(d_census_cost_r);
    hipFree(d_adcensus_cost_l);
    hipFree(d_adcensus_cost_r);
    for (int d = 0; d < num_disp; ++d)
    {
        hipFree(h_ad_cost_l[d]);
        hipFree(h_ad_cost_r[d]);
        hipFree(h_census_cost_l[d]);
        hipFree(h_census_cost_r[d]);
        hipFree(h_adcensus_cost_l[d]);
        hipFree(h_adcensus_cost_r[d]);
    }

    // Host De-allocation
    free(h_ad_cost_l); 
    free(h_ad_cost_r); 
    free(h_census_cost_l); 
    free(h_census_cost_r); 
    free(h_adcensus_cost_l); 
    free(h_adcensus_cost_r); 
}

#endif
